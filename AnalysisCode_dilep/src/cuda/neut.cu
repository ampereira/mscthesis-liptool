#include "hip/hip_runtime.h"
#include "neut.cuh"
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>

using std::vector;
using namespace std;

		unsigned GRID_SIZE;
		unsigned BLOCK_SIZE;
		unsigned NUM_THREADS;

namespace Dilep {
	namespace GPU {
		__device__
		double gaus_kernel (double mean, double sigma, hiprandStateMtgp32 *state) {
			// Samples a random number from the standard Normal (Gaussian) Distribution 
			// with the given mean and sigma.                                                 
			// Uses the Acceptance-complement ratio from W. Hoermann and G. Derflinger 
			// This is one of the fastest existing method for generating normal random variables. 
			// It is a factor 2/3 faster than the polar (Box-Muller) methused in the previous 
			// version of TRandom::Gaus. The speed is comparable to the Ziggurat method (from Marsaglia)
			// implemented for example in GSL and available in the MathMore library. 
			//                                                                           
			// REFERENCE:  - W. Hoermann and G. Derflinger (1990):                       
			//              The ACR Method for generating normal random variables,       
			//              OR Spektrum 12 (1990), 181-185.                             
			//                                                                           
			// Implementation taken from 
			// UNURAN (c) 2000  W. Hoermann & J. Leydold, Institut f. Statistik, WU Wien 
			
			unsigned tid = threadIdx.x + blockIdx.x * blockDim.x;

			const double kC1 = 1.448242853;
			const double kC2 = 3.307147487;
			const double kC3 = 1.46754004;
			const double kD1 = 1.036467755;
			const double kD2 = 5.295844968;
			const double kD3 = 3.631288474;
			const double kHm = 0.483941449;
			const double kZm = 0.107981933;
			const double kHp = 4.132731354;
			const double kZp = 18.52161694;
			const double kPhln = 0.4515827053;
			const double kHm1 = 0.516058551;
			const double kHp1 = 3.132731354;
			const double kHzm = 0.375959516;
			const double kHzmp = 0.591923442;

			const double kAs = 0.8853395638;
			const double kBs = 0.2452635696;
			const double kCs = 0.2770276848;
			const double kB  = 0.5029324303;
			const double kX0 = 0.4571828819;
			const double kYm = 0.187308492 ;
			const double kS  = 0.7270572718 ;
			const double kT  = 0.03895759111;

			double result;
			double rn,x,y,z;


			do{
		         y = cuRand(&state[blockIdx.x]);

		         if (y>kHm1) {
		             result = kHp*y-kHp1; break; }

		         else if (y<kZm) {
		             rn = kZp*y-1;
		             result = (rn>0) ? (1+rn) : (-1+rn);
		             break;
		         }

		         else if (y<kHm) {
		             rn = cuRand(&state[blockIdx.x]);
		             rn = rn-1+rn;
		             z = (rn>0) ? 2-rn : -2-rn;
		             if ((kC1-y)*(kC3+abs(z))<kC2) {
		                 result = z; break; }
		             else {
		                 x = rn*rn;
		                 if ((y+kD1)*(kD3+x)<kD2) {
		                     result = rn; break; }
		                 else if (kHzmp-y<exp(-(z*z+kPhln)/2)) {
		                     result = z; break; }
		                 else if (y+kHzm<exp(-(x+kPhln)/2)) {
		                     result = rn; break; }
		             }
		         }

		         while (1) {
		             x = cuRand(&state[blockIdx.x]);
		             y = kYm * cuRand(&state[blockIdx.x]);
		             z = kX0 - kS*x - y;
		             if (z>0)
		                 rn = 2+y/x;
		             else {
		                 x = 1-x;
		                 y = kYm-y;
		                rn = -(2+y/x);
		             }
		             if ((y-kAs+x)*(kCs+x)+kBs<0) {
		                 result = rn; break; }
		             else if (y<x+kT)
		                 if (rn*rn<4*(kB-log(x))) {
		                     result = rn; break; }
		         }
   } while(0);

			return mean + sigma * result;
		}

		__host__ 
		void gpu_init (int blocks, int threads) {
			GRID_SIZE=blocks;
			BLOCK_SIZE=threads;
			NUM_THREADS=blocks*threads;
		}
	
		__host__
		double calcMass (double x, double y, double z, double e) {
			double mm, mass;

			mm = e*e - (x*x + y*y + z*z);

			if (mm < 0.0)
				mass = -sqrt(-mm);
			else
				mass = sqrt(mm);

			return mass;
		}

		// Wrapper for the dilep calculation using the input class
		__host__
		void dilep (DilepInput &di) {
			std::vector<myvector> *result = new std::vector<myvector> ();
			int hasSolution = 0;

			// time measurement
			#ifdef MEASURE_DILEP
			long long int time = startTimer();
			#endif

			double in_mpx[2], in_mpy[2], in_mpz[2], t_mass[2], w_mass[2];
			TLorentzVector lep_a, lep_b, bl_a, bl_b;

			// Transferring the inputs to local variables
			in_mpx[0] = di.getInMpx(0);
			in_mpx[1] = di.getInMpx(1);
			in_mpy[0] = di.getInMpy(0);
			in_mpy[1] = di.getInMpy(1);
			in_mpz[0] = di.getInMpz(0);
			in_mpz[1] = di.getInMpz(1);
			t_mass[0] = di.getTmass(0);
			t_mass[1] = di.getTmass(1);
			w_mass[0] = di.getWmass(0);
			w_mass[1] = di.getWmass(1);

			lep_a = di.getZlep();
			lep_b = di.getClep();
			bl_a = di.getZbl();
			bl_b = di.getCbl();

			//result = calc_dilep(t_mass, w_mass, in_mpx, in_mpy, in_mpz, &lep_a, &lep_b, &bl_a, &bl_b);

			// Check if there is any solutions for this reconstruction
			if (result->size())
				++hasSolution;  // increment solution counter

			di.setHasSol(hasSolution);
			di.setResult(result);

			// time measurement
			#ifdef MEASURE_DILEP
			stopTimer(time);
			#endif
		}

		// Wrapper for the dilep calculation using a vector of the input class
		// vdi vector with DilepInput varied for a jet combo
		__host__
		void dilep (vector<DilepInput> &vdi) {
			
			double in_mpx[2 * vdi.size()], in_mpy[2 * vdi.size()], in_mpz[2 * vdi.size()], 
				   t_mass[2 * vdi.size()], w_mass[2 * vdi.size()];
			
			double *dev_t_mass, *dev_w_mass, *dev_in_mpx, *dev_in_mpy, *dev_in_mpz;
			double a[5 * vdi.size()], b[5 * vdi.size()], c[5 * vdi.size()], d[5 * vdi.size()];
		
			double *dev_lep_a, *dev_lep_b, *dev_bl_a, *dev_bl_b;
			double nc[16*NUM_THREADS];
			double dev_nc[16*NUM_THREADS];
			int count[NUM_THREADS], dev_count[16*NUM_THREADS];
			int hasSolution = 0;

			// time measurement
			#ifdef MEASURE_DILEP
			long long int time = startTimer();
			#endif

			for (unsigned i = 0; i < vdi.size(); ++i) {

				in_mpx[i * 2]		= vdi[i].getInMpx(0);
				in_mpx[(i * 2) + 1] = vdi[i].getInMpx(1);
				in_mpy[i * 2]		= vdi[i].getInMpy(0);
				in_mpy[(i * 2) + 1] = vdi[i].getInMpy(1);
				in_mpz[i * 2]		= vdi[i].getInMpz(0);
				in_mpz[(i * 2) + 1] = vdi[i].getInMpz(1);
				t_mass[i * 2]		= vdi[i].getTmass(0);
				t_mass[(i * 2) + 1] = vdi[i].getTmass(1);
				w_mass[i * 2]		= vdi[i].getWmass(0);
				w_mass[(i * 2) + 1] = vdi[i].getWmass(1);
					
				a[i * 5]	   = vdi[i].getZlep().Px();
				a[(i * 5) + 1] = vdi[i].getZlep().Py();
				a[(i * 5) + 2] = vdi[i].getZlep().Pz();
				a[(i * 5) + 3] = vdi[i].getZlep().E();
				a[(i * 5) + 4] = vdi[i].getZlep().M();

				b[i * 5]	   = vdi[i].getClep().Px();
				b[(i * 5) + 1] = vdi[i].getClep().Py();
				b[(i * 5) + 2] = vdi[i].getClep().Pz();
				b[(i * 5) + 3] = vdi[i].getClep().E();
				b[(i * 5) + 4] = vdi[i].getClep().M();

				c[i * 5]	   = vdi[i].getZbl().Px();
				c[(i * 5) + 1] = vdi[i].getZbl().Py();
				c[(i * 5) + 2] = vdi[i].getZbl().Pz();
				c[(i * 5) + 3] = vdi[i].getZbl().E();
				c[(i * 5) + 4] = vdi[i].getZbl().M();

				d[i * 5]	   = vdi[i].getCbl().Px();
				d[(i * 5) + 1] = vdi[i].getCbl().Py();
				d[(i * 5) + 2] = vdi[i].getCbl().Pz();
				d[(i * 5) + 3] = vdi[i].getCbl().E();
				d[(i * 5) + 4] = vdi[i].getCbl().M();
			}
/*		if((
				in_mpx[0] == vdi[0].getInMpx(0) &&
				in_mpx[1] == vdi[0].getInMpx(1) &&
				in_mpy[0] == vdi[0].getInMpy(0) &&
				in_mpy[1] == vdi[0].getInMpy(1) &&
				in_mpz[0] == vdi[0].getInMpz(0) &&
				in_mpz[1] == vdi[0].getInMpz(1) &&
				t_mass[0] == vdi[0].getTmass(0) &&
				t_mass[1] == vdi[0].getTmass(1) &&
				w_mass[0] == vdi[0].getWmass(0) &&
				w_mass[1] == vdi[0].getWmass(1) &&
				
				a[0]	   == vdi[0].getZlep().Px() &&
				a[1] == vdi[0].getZlep().Py() &&
				a[2] == vdi[0].getZlep().Pz() &&
				a[3] == vdi[0].getZlep().E() &&
				a[4] == vdi[0].getZlep().M() &&

				b[0]	   == vdi[0].getClep().Px() &&
				b[1] == vdi[0].getClep().Py() &&
				b[2] == vdi[0].getClep().Pz() &&
				b[3] == vdi[0].getClep().E() &&
				b[4] == vdi[0].getClep().M() &&

				c[0]	   == vdi[0].getZbl().Px() &&
				c[1] == vdi[0].getZbl().Py() &&
				c[2] == vdi[0].getZbl().Pz() &&
				c[3] == vdi[0].getZbl().E() &&
				c[4] == vdi[0].getZbl().M() &&

				d[0]	   == vdi[0].getCbl().Px() &&
				d[1] == vdi[0].getCbl().Py() &&
				d[2] == vdi[0].getCbl().Pz() &&
				d[3] == vdi[0].getCbl().E() &&
				d[4] == vdi[0].getCbl().M()
				)) {
				ofstream of ("lawl.txt", fstream::app);
				of << "falhou" << endl;
				of.close();
}*/
				
/*
			// GPU memory allocation of the inputs and outputs of the dilep kernel
			hipMalloc(&dev_t_mass, vdi.size()*2*sizeof(double));
			hipMalloc(&dev_w_mass, vdi.size()*2*sizeof(double));
			hipMalloc(&dev_in_mpx, vdi.size()*2*sizeof(double));
			hipMalloc(&dev_in_mpy, vdi.size()*2*sizeof(double));
			hipMalloc(&dev_in_mpz, vdi.size()*2*sizeof(double));

			hipMalloc(&dev_lep_a, vdi.size()*sizeof(a));
			hipMalloc(&dev_lep_b, vdi.size()*sizeof(b));
			hipMalloc(&dev_bl_a, vdi.size()*sizeof(c));
			hipMalloc(&dev_bl_b, vdi.size()*sizeof(d));
			
			// allocation of the results
			//FALTA VARIACOES
			hipMalloc(&dev_nc, 16*vdi.size()*sizeof(double));
			hipMalloc(&dev_count, vdi.size()*sizeof(int));


			// transfer the inputs to GPU memory
			hipMemcpy(dev_t_mass, t_mass, 2*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(dev_w_mass, w_mass, 2*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(dev_in_mpx, in_mpx, 2*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(dev_in_mpy, in_mpy, 2*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(dev_in_mpz, in_mpz, 2*sizeof(double), hipMemcpyHostToDevice);

			hipMemcpy(dev_lep_a, &a, vdi.size()*sizeof(a), hipMemcpyHostToDevice);
			hipMemcpy(dev_lep_b, &b, vdi.size()*sizeof(b), hipMemcpyHostToDevice);
			hipMemcpy(dev_bl_a, &c, vdi.size()*sizeof(c), hipMemcpyHostToDevice);
			hipMemcpy(dev_bl_b, &d, vdi.size()*sizeof(d), hipMemcpyHostToDevice);

			// define the dimensions of the grid and blocks
			// i.e. the number of times dilep is executed
			dim3 dimGrid(GRID_SIZE, 1);
			dim3 dimBlock(BLOCK_SIZE, 1);
*/
			// dilep kernel call
			//calc_dilep<<<dimGrid,dimBlock>>>(
			//		dev_t_mass, dev_w_mass, dev_in_mpx, dev_in_mpy, dev_in_mpz, 
			//		dev_lep_a, dev_lep_b, dev_bl_a, dev_bl_b, dev_nc, dev_count);
			int ja = 16 * NUM_THREADS;


			for (int i = 0; i < ja; ++i)
			{
				dev_nc[i] = -1;
				dev_count = -1;
			}

			calc_dilep(t_mass, w_mass, in_mpx, in_mpy, in_mpz, 
					a, b, c, d, dev_nc, dev_count);
				
			//	ofstream of ("lawl2.txt", fstream::app);
			//	of << dev_count[0] << endl;
			//	of.close();

			// A TESTAR SE O SET A -1 RESULTA

			// memory transfer of the results from the GPU
			//FALTA VARIACOES
			//hipMemcpy(nc, dev_nc, 16*vdi.size()*sizeof(double), hipMemcpyDeviceToHost);
			//hipMemcpy(count, dev_count, vdi.size()*sizeof(int), hipMemcpyDeviceToHost);

			// reconstruction of the normal output of dilep
			// o num de combs*vars e o num de threads
			for (unsigned comb = 0; comb < vdi.size(); ++comb) {
				vector<myvector> result;

				for (int sol = 0 ; sol < dev_count[comb] && sol<4 ; sol++) {
					myvector *mv = new myvector( 
						TO1D(dev_nc,comb,sol,0),
						TO1D(dev_nc,comb,sol,1),
						TO1D(dev_nc,comb,sol,2),
						TO1D(dev_nc,comb,sol,3) );
					
					result.push_back(*mv);
				}

				if (result.size()) {
					++hasSolution;  // increment solution counter
				}
				vdi[comb].setHasSol(hasSolution);
				vdi[comb].setResult(&result);
			}

			// frees the memory allocated on GPU
			/*hipFree(dev_t_mass);
			hipFree(dev_w_mass);
			hipFree(dev_in_mpx);
			hipFree(dev_in_mpy);
			hipFree(dev_in_mpz);

			hipFree(dev_lep_a);
			hipFree(dev_lep_b);
			hipFree(dev_bl_a);
			hipFree(dev_bl_b);

			hipFree(dev_count);
			hipFree(dev_nc);*/

			// time measurement
			#ifdef MEASURE_DILEP
			stopTimer(time);
			#endif
			
		}

		__device__
		void applyVariance () {

		}

		// NEUTRINO SOLUTIONS
		// TLorentzVector are now arrays
		//__global__
		__host__
		void calc_dilep(double t_mass[], double w_mass[], 
				double in_mpx[], double in_mpy[], double in_mpz[], double lep_a[], 
				double lep_b[], double bl_a[], double bl_b[], 
				double nc[], int a[])
		{

			//unsigned tid = threadIdx.x + blockIdx.x * blockDim.x;
			unsigned tid = 1;
			double G_1, G_3;
			double WMass_a, WMass_b, tMass_a, tMass_b;


			WMass_a = w_mass[0];
			tMass_a = t_mass[0];
			WMass_b = w_mass[1];
			tMass_b = t_mass[1];  
			
			G_1 = (WMass_a - lep_a[4]) * (WMass_a + lep_a[4]);
			G_3 = (WMass_b - lep_b[4]) * (WMass_b + lep_b[4]);

			double G_5,G_6,G_7,G_8,G_9,G_10,G_11,G_12;
			G_5 = ( bl_a[0]/bl_a[3] - lep_a[0]/lep_a[3] );
			G_6 = ( bl_a[1]/bl_a[3] - lep_a[1]/lep_a[3] );
			G_7 = ( bl_a[2]/bl_a[3] - lep_a[2]/lep_a[3] );
			G_8 = ( G_1/lep_a[3] - ((tMass_a - bl_a[4]) * (tMass_a + bl_a[4]))/bl_a[3] )/2.;

			G_9 =	( bl_b[0]/bl_b[3] - lep_b[0]/lep_b[3] );
			G_10 =	( bl_b[1]/bl_b[3] - lep_b[1]/lep_b[3] );
			G_11 =	( bl_b[2]/bl_b[3] - lep_b[2]/lep_b[3] );
			G_12 =	( G_3/lep_b[3] - ((tMass_b - bl_b[4]) * (tMass_b + bl_b[4]))/bl_b[3] )/2.;

			///////////////////////////////////////////////////////////////////
			//// 	G_5 *x1 + G_6*y1 + G_7*z1 = G8;  		(6)
			////  	G_9 *x2 + G_10*y2 + G_11*z2 = G12; 		(7)
			////  	2*El_1*sqrt() - 2*(ax1+by1+cz1) = G_1;  	(8)
			////  	2*El_2*sqrt() - 2*(Ax2+By2+Cz2) = G_3;		(9)
			////  	x1+x2 = S;					(10)
			////  	y1+y2 = T;					(11)
			////  	bring z1 and z2 (from 6/7) to 7 and 8
			///////////////////////////////////////////////////////////////////

			//// 1st top decay product /////
			
			double in_a[5],out_a[6];
			in_a[0] = G_8/G_7;
			in_a[1] = -1.0*G_5/G_7;
			in_a[2] = -1.0*G_6/G_7;
			in_a[3] = lep_a[3];
			in_a[4] = G_1;
			toz(in_a, lep_a, out_a);

			double in_c[5],out_c[6];
			in_c[0] = G_12/G_11;
			in_c[1] = -1*G_9/G_11;
			in_c[2] = -1*G_10/G_11;
			in_c[3] = lep_b[3];
			in_c[4] = G_3;
			toz(in_c, lep_b, out_c);
			/////////////////////////////////////////////////////
			//////change x2 y2 equation to x1 and y1 by using
			////// 		x1+x2 = S = in_mpx[0]
			////// 		y1+y2 = T = mpy
			/////////////////////////////////////////////////////
			double out_e[6];
			out_e[0] = out_c[0];
			out_e[1] = out_c[1];
			out_e[2] = -1*( out_c[0]*in_mpx[0] + out_c[2] + out_c[4]*in_mpy[0]);
			out_e[3] = -1*( out_c[1]*in_mpy[0] + out_c[3] + out_c[4]*in_mpx[0]);
			out_e[4] = out_c[4]; 
			out_e[5] =( out_c[0]*in_mpx[0]*in_mpx[0] + out_c[1]*in_mpy[0]*in_mpy[0] + 2*out_c[2]*in_mpx[0] + 2*out_c[3]*in_mpy[0] + out_c[5] + 2*out_c[4]*in_mpx[0]*in_mpy[0]);

			///////////////////////////////////////////////////
			///  solve 
			/// {ax2+by2+2dx+2ey+2fxy+g=0		(12)
			/// {Ax2+By2+2Dx+2Ey+2Fxy+G=0		(13)
			/// out_a[6]: 0   1    2    3    4     5
			/// out_a[6]: a   b    d    e    f     g
			/// out_e[6]: A   B    D    E    F     G
			/// if a!=0, everything is OK.
			///
			/// if a==0, then we can get x2 = f(x,y) from (13)
			/// (12) --> [x2 - f(x,y)] + by2 + ... = 0
			///////////////////////////////////////////////////
			
			double fx_1, fx_2, fx_3, fx_4, fx_5;
			double k_1, k_2, k_3, k_4, k_5;

			// bad organization of the code; diminished register spilling
			if ( out_a[0]!=0  ){
				fx_1 = 2.*(out_e[0]*out_a[2] - out_a[0]*out_e[2]);
				fx_2 = 2.*(out_e[0]*out_a[4] - out_a[0]*out_e[4]);
				fx_3 = out_a[0]*out_e[1] - out_e[0]*out_a[1];
				fx_4 = 2.*(out_e[3]*out_a[0] - out_e[0]*out_a[3]);
				fx_5 = out_a[0]*out_e[5] - out_a[5]*out_e[0];

				k_1 = ( out_a[4]*out_a[4] - out_a[0]*out_a[1] )/out_a[0]/out_a[0];
				k_2 = ( 2.*out_a[2]*out_a[4] - 2.*out_a[0]*out_a[3] )/out_a[0]/out_a[0];
				k_3 = ( out_a[2]*out_a[2]-out_a[0]*out_a[5] )/out_a[0]/out_a[0];
				k_4 = -out_a[2]/out_a[0];
				k_5 = -out_a[4]/out_a[0];
			} else {
				if (out_a[0]==0 && out_e[0]!=0 ) {
					fx_1 = 2.*(out_e[0]*(out_a[2] + out_e[2]/out_e[0]) - out_e[2]);
					fx_2 = 2.*(out_e[0]*(out_a[4] + out_e[4]/out_e[0]) - out_e[4]);
					fx_3 = out_e[1] - out_e[0]*(out_a[1] + out_e[1]/out_e[0]);
					fx_4 = 2.*(out_e[3] - out_e[0]*(out_a[3] + out_e[3]/out_e[0]));
					fx_5 = out_e[5] - (out_a[5] + out_e[5]/out_e[0])*out_e[0];

					k_1 = ( (out_a[4] + out_e[4]/out_e[0])*(out_a[4] + out_e[4]/out_e[0]) - (out_a[1] + out_e[1]/out_e[0]) );
					k_2 = ( 2.*(out_a[2] + out_e[2]/out_e[0])*(out_a[4] + out_e[4]/out_e[0]) - 2.*(out_a[3] + out_e[3]/out_e[0]) );
					k_3 = ( (out_a[2] + out_e[2]/out_e[0])*(out_a[2] + out_e[2]/out_e[0])-(out_a[5] + out_e[5]/out_e[0]) );
					k_4 = -(out_a[2] + out_e[2]/out_e[0]);
					k_5 = -(out_a[4] + out_e[4]/out_e[0]);
				}
			}

			if ( out_a[0]==0 && out_e[0]==0){
				return;
			}


			/////
			///// the part above is 
			///// x = (fx3*y**2 + fx4*y + fx5)/(fx1 + fx2*y)
			///// used to get x value once y is known
			/////
			//// if fx1 + fx2*y == 0, then x is 
			//// x = +/-sqrt(k1*y**2 + k2*y + k3) + (k4 + k5*y)
			////


			double g_1 = 4.*out_e[0]*out_e[0]*k_5*k_5 + 4.*out_e[4]*out_e[4] + 8.*out_e[0]*out_e[4]*k_5;
			double m_1 = g_1*k_1;
			double g_2 = 8.*out_e[0]*out_e[0]*k_4*k_5 + 8.*out_e[0]*out_e[2]*k_5 + 8.*out_e[0]*out_e[4]*k_4 + 8.*out_e[2]*out_e[4];
			double g_3 = 4.*out_e[0]*out_e[0]*k_4*k_4 + 4.*out_e[2]*out_e[2] + 8.*out_e[0]*out_e[2]*k_4;
			double g_4 = out_e[0]*k_1 + out_e[0]*k_5*k_5;
			double g_5 = out_e[0]*k_2 + 2.*out_e[0]*k_4*k_5 + 2.*out_e[2]*k_5;
			double g_6 = out_e[0]*k_3 + out_e[0]*k_4*k_4 + 2.*out_e[2]*k_4 + out_e[5];

			double m_2 = g_1*k_2 + g_2*k_1;
			double m_3 = g_1*k_3 + g_2*k_2 + g_3*k_1;
			double m_4 = g_2*k_3 + g_3*k_2;
			double m_5 = g_3*k_3;

			double m_6  = out_e[1]*out_e[1] + 4.*out_e[4]*out_e[4]*k_5*k_5 + 4.*out_e[1]*out_e[4]*k_5;
			double m_7  = 4.*out_e[1]*out_e[3] + 8.*out_e[4]*out_e[4]*k_4*k_5 + 4.*out_e[1]*out_e[4]*k_4 + 8.*out_e[3]*out_e[4]*k_5;
			double m_8  = 4.*out_e[3]*out_e[3] + 4.*out_e[4]*out_e[4]*k_4*k_4 + 8.*out_e[3]*out_e[4]*k_4;
			double m_80 = pow(g_4,2);
			double m_81 = 2*g_4*g_5;
			double m_9  = pow(g_5,2) + 2.*g_4*g_6;
			double m_10 = 2.*g_5*g_6;
			double m_11 = g_6*g_6;

			double m_12 = 	2.*out_e[0]*out_e[1]*k_1 + 2.*out_e[0]*out_e[1]*k_5*k_5 + 4.*out_e[0]*out_e[4]*k_1*k_5 + 4.*out_e[0]*out_e[4]*pow(k_5,3);
			double m_13 = 	2.*out_e[0]*out_e[1]*k_2 + 4.*out_e[0]*out_e[1]*k_4*k_5 + 4.*out_e[1]*out_e[2]*k_5 + 
				4.*out_e[0]*(out_e[3]*k_1 + out_e[3]*k_5*k_5 + out_e[4]*k_1*k_4 + out_e[4]*k_2*k_5) + 
				12.*out_e[0]*out_e[4]*k_4*k_5*k_5 + 8.*out_e[2]*out_e[4]*k_5*k_5;
			double m_14 = 	2.*out_e[0]*out_e[1]*k_3 + 2.*out_e[0]*out_e[1]*k_4*k_4 + 4.*out_e[2]*out_e[1]*k_4 + 2.*out_e[1]*out_e[5] + 4.*out_e[0]*out_e[3]*k_2 + 
				8.*out_e[0]*out_e[3]*k_4*k_5 + 8.*out_e[3]*out_e[2]*k_5 + 4.*out_e[0]*out_e[4]*k_2*k_4 + 4.*out_e[0]*out_e[4]*k_3*k_5 + 
				12.*out_e[0]*out_e[4]*k_4*k_4*k_5 + 16.*out_e[2]*out_e[4]*k_4*k_5 + 4.*out_e[4]*out_e[5]*k_5;
			double m_15 = 	4.*out_e[0]*out_e[3]*(k_3 + k_4*k_4) + 8.*out_e[3]*out_e[2]*k_4 + 4.*out_e[3]*out_e[5] + 4.*out_e[0]*out_e[4]*(k_3*k_4 + pow(k_4,3)) + 
				8.*out_e[2]*out_e[4]*k_4*k_4 + 4.*out_e[4]*out_e[5]*k_4;

			double  re[5];
			re[0] = m_1 - m_6 - m_12 - m_80;
			re[1] = m_2 - m_7 - m_13 - m_81;
			re[2] = m_3 - m_8 - m_9 - m_14;
			re[3] = m_4 - m_10 - m_15;
			re[4] = m_5 - m_11;  



			double output[8];
			my_qu(re, output);

			int ncand(0);

			double rec_x1, rec_y1, rec_z1, rec_e1, rec_x2, rec_y2, rec_z2, rec_e2;

			for (int j=0; j<8; j+=2){
				double delta = k_1*output[j]*output[j] + k_2*output[j] + k_3;
				if ( output[j+1]==0 && delta >=0) {
					if ( (fx_1 + fx_2*output[j])!=0 ) {
						rec_x1 = (fx_3*pow(output[j],2) + fx_4*output[j] + fx_5)/(fx_1 + fx_2*output[j]);
					} else {
						rec_x1 = sqrt(delta)+k_4+k_5*output[j];
					}  

					rec_y1 = output[j];
					rec_z1 = G_8/G_7 - G_5*rec_x1/G_7 - G_6*rec_y1/G_7;
					rec_e1 = sqrt(rec_x1*rec_x1 + rec_y1*rec_y1 + rec_z1*rec_z1);
					rec_x2 = in_mpx[0] - rec_x1;
					rec_y2 = in_mpy[0] - rec_y1;
					rec_z2 = G_12/G_11 - G_9*rec_x2/G_11 - G_10*rec_y2/G_11;
					rec_e2 = sqrt(rec_x2*rec_x2 + rec_y2*rec_y2 + rec_z2*rec_z2);
					
					// self-consistence check and control of the solutions

					double m_w11 = calcMass(rec_x1+lep_a[0], rec_y1+lep_a[1], rec_z1+lep_a[2], rec_e1+lep_a[3]);
					double m_w12 = calcMass(rec_x2+lep_b[0], rec_y2+lep_b[1], rec_z2+lep_b[2], rec_e2+lep_b[3]);
					double m_t11 = calcMass(rec_x1+ bl_a[0], rec_y1+ bl_a[1], rec_z1+ bl_a[2], rec_e1+ bl_a[3]);
					double m_t12 = calcMass(rec_x2+ bl_b[0], rec_y2+ bl_b[1], rec_z2+ bl_b[2], rec_e2+ bl_b[3]);

					// m_delta_mass is 1000.0
					bool m_good_eq1 = ( fabs(in_mpx[0] -(rec_x1+rec_x2)) <= 0.01 ) * true + 
									  ( fabs(in_mpx[0] -(rec_x1+rec_x2)) > 0.01 ) * false;
					bool m_good_eq2 = ( fabs(in_mpy[0] -(rec_y1+rec_y2)) <= 0.01 ) * true +
									  ( fabs(in_mpy[0] -(rec_y1+rec_y2)) > 0.01 ) * false;
					bool m_good_eq3 = ( fabs(m_w11 - w_mass[0]) <= 1000.0 ) * true + 
									  ( fabs(m_w11 - w_mass[0]) > 1000.0 ) * false;
					bool m_good_eq4 = ( fabs(m_w12 - w_mass[1]) <= 1000.0 ) * true +
									  ( fabs(m_w12 - w_mass[1]) > 1000.0 ) * false;
					bool m_good_eq5 = ( fabs(m_t11 - t_mass[0]) <= 1000.0 ) * true +
									  ( fabs(m_t11 - t_mass[0]) > 1000.0 ) * false;
					bool m_good_eq6 = ( fabs(m_t12 - t_mass[1]) <= 1000.0 ) * true +
									  ( fabs(m_t12 - t_mass[1]) <= 1000.0 ) * false;

					bool cond = m_good_eq1 && m_good_eq2 && m_good_eq3 && m_good_eq4 && m_good_eq5 && m_good_eq6;
					
					// aqui podem nao chegar as threads todas
					//__syncthreads();
					nc[tid * 16 + 2*j] = cond * rec_x1;
					nc[tid * 16 + 2*j + 1] = cond * rec_y1;
					nc[tid * 16 + 2*j + 2] = cond * rec_z1;
					nc[tid * 16 + 2*j + 3] = cond * rec_z2;
					ncand += cond * 1;
				}
			}

			// indicates the number of solutions that this thread found
			a[tid] = ncand;
		}

		//////////////////////////////////////
		__host__
		void toz(double k[], double l[], double g[]){
			//// checked !!
			///////////////////////////////////////////////////////////////////////////
			///// bring z=A+Bx+Cy to 2*D*sqrt(x**2+y**2+z**2)-2(ax+by+dz) = E
			///// simplify it to g1*x^2 + g2*y^2 + 2*g3*x + 2*g4*y + 2*g5*xy + g6 = 0
			///////////////////////////////////////////////////////////////////////////
			double A = k[0];
			double B = k[1];
			double C = k[2];
			double D = k[3];
			double E = k[4];
			double a = l[0];
			double b = l[1];
			double d = l[2];
			g[0] = 4*pow(D,2)*( 1 + pow(B,2)) - 4*pow(a,2) -4*pow(d,2)*pow(B,2) - 8*a*d*B;
			if ( g[0]!=0 ) { 
				g[1] = ( 4*pow(D,2)*( 1 + pow(C,2)) - 4*pow(b,2) -4*pow(d,2)*pow(C,2) - 8*b*d*C )/g[0] ;
				g[2] = ( (4*pow(D,2)-4*d*d)*A*B - 4*a*d*A - 2*E*a - 2*E*d*B )/g[0];
				g[3] = ( (4*pow(D,2)-4*d*d)*A*C - 4*b*d*A - 2*E*b - 2*E*d*C )/g[0];
				g[4] = ( (4*pow(D,2)-4*d*d)*B*C - 4*a*b   - 4*a*d*C - 4*b*d*B )/g[0];
				g[5] = ( (4*pow(D,2)-4*d*d)*A*A - E*E - 4*E*d*A )/g[0];
				g[0] = 1.0; 
			} else {
				g[1] = ( 4*pow(D,2)*( 1 + pow(C,2)) - 4*pow(b,2) -4*pow(d,2)*pow(C,2) - 8*b*d*C ) ;
				g[2] = ( (4*pow(D,2)-4*d*d)*A*B - 4*a*d*A - 2*E*a - 2*E*d*B );
				g[3] = ( (4*pow(D,2)-4*d*d)*A*C - 4*b*d*A - 2*E*b - 2*E*d*C );
				g[4] = ( (4*pow(D,2)-4*d*d)*B*C - 4*a*b   - 4*a*d*C - 4*b*d*B );
				g[5] = ( (4*pow(D,2)-4*d*d)*A*A - E*E - 4*E*d*A );
				g[0] = 0.;  
			}	 
			return;
		}


		///////////////////////////////////////////
		__host__
		void my_qu( double my_in[], double my_val[])
		{

			///////////////////////////////////////////
			/////  find the solution /////////////////
			/////  ax^4+bx^3+cx^2+dx+e=0
			//////////////////////////////////////////
			double a=my_in[0];
			double b=my_in[1];
			double c=my_in[2];
			double d=my_in[3];
			double e=my_in[4];

			double real[3]={0,0,0};
			double img[3]={0,0,0};
			double x1_r = 0; double x1_i = 0;
			double x2_r = 0; double x2_i = 0;
			double x3_r = 0; double x3_i = 0;
			double x4_r = 0; double x4_i = 0;


			/////////////////////////////////////////////
			///// in case of a==0, simplify to cubic
			///// bx^3+cx^2+dx+e=0
			/////////////////////////////////////////////

			if ( a ==0 && b!=0 ){
				double input[4]={b,c,d,e};
				cubic(input,real,img);
				x1_r = real[0];	x1_i = img[0];
				x2_r = real[1];	x2_i = img[1];
				x3_r = real[2];	x3_i = img[2];
				my_val[0] = x1_r;
				my_val[1] = x1_i;
				my_val[2] = x2_r;
				my_val[3] = x2_i;
				my_val[4] = x3_r;
				my_val[5] = x3_i;
				my_val[6] = x4_r;
				my_val[7] = x4_i;
				
				return;
			}


			/////////////////////////////////////////////
			///// in case of a==0 && b==0, 
			///// simplify to quadratic
			///// cx*2 + d*x + e =0
			/////////////////////////////////////////////

			if ( a ==0 && b==0 && c!=0){
				double alpha = pow(d,2)-4*c*e;
				if (alpha>=0) {
					x1_r = (-1*d + sqrt(alpha))/2/c;	x1_i = 0;
					x2_r = (-1*d - sqrt(alpha))/2/c;	x2_i = 0;
				} else {
					x1_r = (-1*d )/2/c;	x1_i = sqrt(-alpha)/2/c;
					x2_r = (-1*d )/2/c;	x2_i = -1*sqrt(-alpha)/2/c;  	
				}
				my_val[0] = x1_r;
				my_val[1] = x1_i;
				my_val[2] = x2_r;
				my_val[3] = x2_i;
				my_val[4] = x3_r;
				my_val[5] = x3_i;
				my_val[6] = x4_r;
				my_val[7] = x4_i;
				
				return;
			}


			/////////////////////////////////////////////
			///// in case of a==0 && b==0 && c==0, 
			///// simplify to linear equation dx + e =0
			/////////////////////////////////////////////

			if (a ==0 && b==0 && c==0 && d!=0){
				x1_r = -e/d;	x1_i = 0;
				my_val[0] = x1_r;
				my_val[1] = x1_i;
				my_val[2] = x2_r;
				my_val[3] = x2_i;
				my_val[4] = x3_r;
				my_val[5] = x3_i;
				my_val[6] = x4_r;
				my_val[7] = x4_i;
				
				return;	
			}


			//////////////////////////////////////////////
			////  (1)the common expression
			///////////////////////////////////////////////
			double bb=b/a; double cc=c/a;
			double dd=d/a; double ee=e/a;

			//////////////////////////////////////////////
			////  (2) the equation changes to 
			////	x^4 + bb*x^3 + cc*x^2 + dd*x + ee=0
			////
			////  (3) substitude x=y-aa/4, then we get
			////    y^4 + f*y^2 + g*y + h =0; where
			///////////////////////////////////////////

			double _f = cc - 3*pow(bb,2)/8;
			double _g = dd + (pow(bb,3)/8) - (bb*cc/2);
			double _h = ee - (3*pow(bb,4)/256) + (pow(bb,2)*cc/16) - (bb*dd/4);

			////////////////////////////////////////////////////////////////////////////
			///  (4) the normal situation is f, g and h are non-zero; then		////
			//// 	the related cubic equation is					////
			//// 	z^3 + (f/2) z^2 + ( (f^2-4h)/16 )*z -g^2/64 =0;			////
			////	 it has three "squared" roots, for example, p,q and l, 		////
			//// 	then p^2, q^2 and l^2 are the root of equation above		////
			//// 	set r=-g/8(pq), then the four roots of the original quartic are	////
			//// 		x = p + q + r -bb/4;					////
			//// 		x = p - q - r -bb/4;					////
			//// 		x = -p + q - r -bb/4;					////
			//// 		x = -p - q + r -bb/4;					////
			////////////////////////////////////////////////////////////////////////////

			double c_1 = 1.;
			double c_2 = _f/2;
			double c_3 = (pow(_f,2)-4*_h)/16.;
			double c_4 = -1*pow(_g,2)/64.;

			double input[4]={c_1,c_2,c_3,c_4};
			cubic(input,real,img);

			////////////////////////////////////////////////
			////// (5) sqrt root of the cubic equation solutions
			////////////////////////////////////////////////

			double out_r[3]={0,0,0};
			double out_i[3]={0,0,0};
			bool _img[3]={false,false,false};
			int img_index =0;
			double my[2];

			for (int ii=0; ii<3; ii++){
				Csqrt(real[ii],img[ii],my);
				out_r[ii] = my[0];
				out_i[ii] = my[1];
				if ( my[1]!=0 ) {
					_img[ii]=true;
					img_index++;
				}
			}    


			///////////////////////////////////////////////
			////(x + yi)(u + vi) = (xu - yv) + (xv + yu)i
			//// calculating r = -g/(8pq)
			////////////////////////////////////////////////
			double r_r;	double r_i;
			double p_r;	double p_i;
			double q_r;	double q_i;

			if (_img[0]==_img[1] && out_r[0]*out_r[1]>0 ){
				r_r	= out_r[0]*out_r[1] - out_i[0]*out_i[1];
				r_i	= out_r[0]*out_i[1] + out_r[1]*out_i[0];
				p_r	= out_r[0]; p_i = out_i[0];
				q_r	= out_r[1]; q_i = out_i[1];
			} else {	
				for (int kk=0; kk<2; kk++) {
					for (int k=kk+1; k<3; k++) {
						if (_img[kk]==_img[k]){ 
							r_r	= out_r[kk]*out_r[k] - out_i[kk]*out_i[k];
							r_i	= out_r[kk]*out_i[k] + out_r[k]*out_i[kk];
							p_r	= out_r[kk]; p_i = out_i[kk];
							q_r	= out_r[k]; q_i = out_i[k];
						}
					}
				}
			}


			if( r_r !=0 )	{ 
				r_r 	= -1.*_g/8/r_r;
			} else {
				r_r 	= 0;
			}
			if( r_i !=0 )	{ 
				r_r 	= -1.*_g/8/r_i;
			} else {
				r_i 	= 0;
			}	

			x1_r = p_r + q_r + r_r -b/a/4;
			x1_i = p_i + q_i + r_i;
			x2_r = p_r - q_r - r_r -b/a/4;
			x2_i = p_i - q_i - r_i;
			x3_r = -1*p_r + q_r - r_r -b/a/4;
			x3_i = -1*p_i + q_i - r_i;
			x4_r = -1*p_r - q_r + r_r -b/a/4;
			x4_i = -1*p_i - q_i + r_i;

			my_val[0] = x1_r;
			my_val[1] = x1_i;
			my_val[2] = x2_r;
			my_val[3] = x2_i;
			my_val[4] = x3_r;
			my_val[5] = x3_i;
			my_val[6] = x4_r;
			my_val[7] = x4_i;
		}
		////////////////////end of main
		///////////////////////////////////////////////////////////////
		////+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
		__host__
		void Csqrt(double _ar, double _ai, double _my[])
		{
			///// complex sqrt
			double x,y,r,w;
			if  (  (_ar == 0.0) && (_ai == 0.0) ) {
				_my[0]=0.0;
				_my[1]=0.0;
				return;
			} else {	
				x=fabs(_ar);
				y=fabs(_ai);
				if (x >= y) {
					r=y/x;
					w=sqrt(x)*sqrt(0.5*(1.0+sqrt(1.0+r*r)));
				} else {
					r=x/y;
					w=sqrt(y)*sqrt(0.5*(r+sqrt(1.0+r*r)));
				}
				if (_ar>= 0.0) {
					_my[0]=w;
					_my[1]=_ai/(2.0*w);
				} else {
					_my[1]=(_ai >= 0) ? w : -w;
					_my[0]=_ai/(2.0*_my[1]);
				}
				return;
			}
		}

		//////////////////////////////////////////////////////////////////
		/// cubic /// a[0]x^3+a[1]x^2+a[2]x+a[3]=0
		//////////////////////////////////////////////////////////////////
		__host__
		void cubic(double a[], double rr[], double ri[])
		{
			int i;
			double a0, a1, a2, a3;
			double g, h, y1, sh, theta, pi, xy1, xy2, xy3;
			double y2, z1, z2, z3, z4;
			//// initialize the results
			for (i = 0; i < 3; i ++)
			{
				rr[i] = 0.0;
				ri[i] = 0.0;
			}

			a0 = a[0];
			a1 = a[1]/3.0;
			a2 = a[2]/3.0;
			a3 = a[3];

			g = (a0 * a0) * a3 - 3.0 * a0 * a1 * a2 + 2.0 * pow(a1, 3);
			h = a0 * a2 - a1 * a1;
			y1 = g * g + 4.0 * pow(h, 3);

			if (y1 < 0.0){
				sh = sqrt(-h);
				theta = acos(g / (2.0 * h * sh)) / 3.0;
				xy1 = 2.0 * sh * cos(theta);				xy2 = 2.0 * sh * cos(theta + (2.0 * PI / 3.0));
				xy3 = 2.0 * sh * cos(theta + (4.0 * PI / 3.0));
				rr[0] = (xy1 - a1) / a0;
				rr[1] = (xy2 - a1) / a0;
				rr[2] = (xy3 - a1) / a0;
				return;
			} else {
				y2 = sqrt(y1);
				z1 = (g + y2) / 2.0;
				z2 = (g - y2) / 2.0;
				if (z1 < 0.0){
					z3 = pow(-z1, 1.0/3.0);
					z3 = -z3;
				} else  z3 = pow(z1, 1.0/3.0);
				if (z2 < 0.0){
					z4 = pow(-z2, 1.0/3.0);
					z4 = - z4;
				}
				else  z4 = pow(z2, 1.0/3.0);

				rr[0] = -(a1 + z3 + z4) / a0;
				rr[1] = (-2.0 * a1 + z3 + z4) / (2.0 * a0);
				ri[1] = sqrt(3.0) * (z4 - z3) / (2.0 * a0);
				rr[2] = rr[1];
				ri[2] = -ri[1];

				return;

			}
		}
	}
}

