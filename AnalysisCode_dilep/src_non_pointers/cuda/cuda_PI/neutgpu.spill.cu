#include "hip/hip_runtime.h"

#include "neutgpu.cuh"
#include <cmath>
// defines para o CUDA

unsigned int GRID_SIZE;		// num de blocos por grid, vai dar 10^6
unsigned int BLOCK_SIZE;	// num threads por bloco

float total_M1, total_M2, total_K;

unsigned int NUM_THREADS; // = (1*8192*128);

__host__ 
void gpu_init(int blocks,int threads)	{
	GRID_SIZE=blocks;
	BLOCK_SIZE=threads;
	NUM_THREADS=blocks*threads;
}

__host__
void printIterations()	{
	printf("%d",NUM_THREADS);
}
/**
 *		Comparacoes de numeros em virgula flutuante sao problematicas, pelo que todas terao associadas
 * um erro relativo de 0.00000
 */

__device__
bool equal ( const double _this, const double &other) 	{
		double tmp = abs(_this-other);
		return ((tmp/abs(_this))< 0.000001);
}

__device__
bool nequal(const double _this, const double &other)	{
	return (!equal(_this,other));
}

__device__
double calcMass(double x, double y, double z, double e){
	double mm, mass;

	mm = e * e - (x * x + y * y + z * z);

	if(mm < 0.0)
		mass = -sqrt(-mm);
	else
		mass = sqrt(mm);

	return mass;
}

// FILE* flog;

// NEUTRINO SOLUTIONS


	__global__
void dilep_kernel(double t_mass[], double w_mass[], double in_mpx[], 
//		double in_mpy[], double in_mpz[], LorentzVector *lep_a, 
		double in_mpy[], double in_mpz[], double lep_a[], 
//		LorentzVector *lep_b, LorentzVector *bl_a, LorentzVector *bl_b, 
		double lep_b[], double bl_a[], double bl_b[], 
		double nc[], int a[])
{

	///////////////////////////////////////////////////////////////////////////////////////////////////////
	// Neste codigo esta-se a assumir que LorentzVector.m contem o valor precalculado de LorentzVector->M()
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	/*_LorentzVectorM(*lep_a);
	_LorentzVectorM(*lep_b);
	_LorentzVectorM(*bl_a);
	_LorentzVectorM(*bl_b);
*/
	unsigned tid = threadIdx.x + blockIdx.x * blockDim.x;
	double mpx, mpy, G_1, G_2, G_3, G_4;
	double WMass_a, WMass_b, tMass_a, tMass_b;

	mpx = in_mpx[0];
	mpy = in_mpy[0];

	WMass_a = w_mass[0];
	tMass_a = t_mass[0];
	WMass_b = w_mass[1];
	tMass_b = t_mass[1];  
	
	// Cancelamento subractivo
	/*
	G_1 = WMass_a*WMass_a - ( lep_a[4] )*( lep_a[4] );
	G_3 = WMass_b*WMass_b - ( lep_b[4] )*( lep_b[4] );
	G_2 = tMass_a*tMass_a - ( bl_a[4]  )*( bl_a[4]  ); 
	G_4 = tMass_b*tMass_b - ( bl_b[4]  )*( bl_b[4]  );
	*/
	
	G_1 = (WMass_a - lep_a[4]) * (WMass_a + lep_a[4]);
	G_3 = (WMass_b - lep_b[4]) * (WMass_b + lep_b[4]);
	G_2 = (tMass_a - bl_a[4]) * (tMass_a + bl_a[4]);
	G_4 = (tMass_b - bl_b[4]) * (tMass_b + bl_b[4]);

	double S=mpx;
	double T=mpy;

	double G_5,G_6,G_7,G_8,G_9,G_10,G_11,G_12;
	G_5 = ( bl_a[0]/bl_a[3] - lep_a[0]/lep_a[3] );
	G_6 = ( bl_a[1]/bl_a[3] - lep_a[1]/lep_a[3] );
	G_7 = ( bl_a[2]/bl_a[3] - lep_a[2]/lep_a[3] );
	G_8 = ( G_1/lep_a[3] - G_2/bl_a[3] )/2.;

	G_9 =	( bl_b[0]/bl_b[3] - lep_b[0]/lep_b[3] );
	G_10 =	( bl_b[1]/bl_b[3] - lep_b[1]/lep_b[3] );
	G_11 =	( bl_b[2]/bl_b[3] - lep_b[2]/lep_b[3] );
	G_12 =	( G_3/lep_b[3] - G_4/bl_b[3] )/2.;

	///////////////////////////////////////////////////////////////////
	//// 	G_5 *x1 + G_6*y1 + G_7*z1 = G8;  		(6)
	////  	G_9 *x2 + G_10*y2 + G_11*z2 = G12; 		(7)
	////  	2*El_1*sqrt() - 2*(ax1+by1+cz1) = G_1;  	(8)
	////  	2*El_2*sqrt() - 2*(Ax2+By2+Cz2) = G_3;		(9)
	////  	x1+x2 = S;					(10)
	////  	y1+y2 = T;					(11)
	////  	bring z1 and z2 (from 6/7) to 7 and 8
	///////////////////////////////////////////////////////////////////

	//// 1st top decay product /////
	
	double in_a[5],on_a[3],out_a[6];
	in_a[0] = G_8/G_7;
	in_a[1] = -1.0*G_5/G_7;
	in_a[2] = -1.0*G_6/G_7;
	in_a[3] = lep_a[3];
	in_a[4] = G_1;

	on_a[0] = lep_a[0];
	on_a[1] = lep_a[1];
	on_a[2] = lep_a[2];
	toz_kernel(in_a, on_a, out_a);

	double in_c[5],on_c[3],out_c[6];
	in_c[0] = G_12/G_11;
	in_c[1] = -1*G_9/G_11;
	in_c[2] = -1*G_10/G_11;
	in_c[3] = lep_b[3];
	in_c[4] = G_3;

	on_c[0] = lep_b[0];
	on_c[1] = lep_b[1];
	on_c[2] = lep_b[2];
	toz_kernel(in_c, on_c, out_c);
	/////////////////////////////////////////////////////
	//////change x2 y2 equation to x1 and y1 by using
	////// 		x1+x2 = S = mpx
	////// 		y1+y2 = T = mpy
	/////////////////////////////////////////////////////
	double out_e[6];
	out_e[0] = out_c[0];
	out_e[1] = out_c[1];
	out_e[2] = -1*( out_c[0]*mpx + out_c[2] + out_c[4]*mpy);
	out_e[3] = -1*( out_c[1]*mpy + out_c[3] + out_c[4]*mpx);
	out_e[4] = out_c[4]; 
	out_e[5] =( out_c[0]*mpx*mpx + out_c[1]*mpy*mpy + 2*out_c[2]*mpx + 2*out_c[3]*mpy + out_c[5] + 2*out_c[4]*mpx*mpy);

	///////////////////////////////////////////////////
	///  solve 
	/// {ax2+by2+2dx+2ey+2fxy+g=0		(12)
	/// {Ax2+By2+2Dx+2Ey+2Fxy+G=0		(13)
	/// out_a[6]: 0   1    2    3    4     5
	/// out_a[6]: a   b    d    e    f     g
	/// out_e[6]: A   B    D    E    F     G
	/// if a!=0, everything is OK.
	///
	/// if a==0, then we can get x2 = f(x,y) from (13)
	/// (12) --> [x2 - f(x,y)] + by2 + ... = 0
	///////////////////////////////////////////////////
	
	double _A = out_e[0];
	double _B = out_e[1];
	double _D = out_e[2];
	double _E = out_e[3];
	double _F = out_e[4];
	double _G = out_e[5];
	
	double _d, _za, _f, _b, _e, _g;
	double fx_1, fx_2, fx_3, fx_4, fx_5;
	double k_1, k_2, k_3, k_4, k_5;
/*
	//if ( out_a[0]!=0  ){
	_a = ( out_a[0]!=0  ) * out_a[0];
	_b = ( out_a[0]!=0  ) * out_a[1];
	_d = ( out_a[0]!=0  ) * out_a[2];
	_e = ( out_a[0]!=0  ) * out_a[3];
	_f = ( out_a[0]!=0  ) * out_a[4];
	_g = ( out_a[0]!=0  ) * out_a[5];

	//if (out_a[0]==0 && _A!=0 ) {
	_a = (out_a[0]==0 && _A!=0) * 1.;
	_b = (out_a[0]==0 && _A!=0) * (out_a[1] + _B/_A);
	_d = (out_a[0]==0 && _A!=0) * (out_a[2] + _D/_A);
	_e = (out_a[0]==0 && _A!=0) * (out_a[3] + _E/_A);
	_f = (out_a[0]==0 && _A!=0) * (out_a[4] + _F/_A);
	_g = (out_a[0]==0 && _A!=0) * (out_a[5] + _G/_A);*/

	if ( out_a[0]!=0  ){
		_za = out_a[0];
		_b = out_a[1];
		_d = out_a[2];
		_e = out_a[3];
		_f = out_a[4];
		_g = out_a[5];

		/* badalhoquice!!! */
		fx_1 = 2.*(_A*_d - _za*_D);
		fx_2 = 2.*(_A*_f - _za*_F);
		fx_3 = _za*_B - _A*_b;
		fx_4 = 2.*(_E*_za - _A*_e);
		fx_5 = _za*_G - _g*_A;

		k_1 = ( _f*_f - _za*_b )/_za/_za;
		k_2 = ( 2.*_d*_f - 2.*_za*_e )/_za/_za;
		k_3 = ( _d*_d-_za*_g )/_za/_za;
		k_4 = -1.*_d/_za;
		k_5 = -1.*_f/_za;
	} else {
		if (out_a[0]==0 && _A!=0 ) {
			_za = 1.;
			_b = out_a[1] + _B/_A;
			_d = out_a[2] + _D/_A;
			_e = out_a[3] + _E/_A;
			_f = out_a[4] + _F/_A;
			_g = out_a[5] + _G/_A;

		/* badalhoquice!!! */
			fx_1 = 2.*(_A*_d - _za*_D);
			fx_2 = 2.*(_A*_f - _za*_F);
			fx_3 = _za*_B - _A*_b;
			fx_4 = 2.*(_E*_za - _A*_e);
			fx_5 = _za*_G - _g*_A;

			k_1 = ( _f*_f - _za*_b )/_za/_za;
			k_2 = ( 2.*_d*_f - 2.*_za*_e )/_za/_za;
			k_3 = ( _d*_d-_za*_g )/_za/_za;
			k_4 = -1.*_d/_za;
			k_5 = -1.*_f/_za;
		}
	}

	if ( out_a[0]==0 && _A==0){
		return;
	}


	/////
	///// the part above is 
	///// x = (fx3*y**2 + fx4*y + fx5)/(fx1 + fx2*y)
	///// used to get x value once y is known
	/////
	//// if fx1 + fx2*y == 0, then x is 
	//// x = +/-sqrt(k1*y**2 + k2*y + k3) + (k4 + k5*y)
	////


	double g_1 = 4.*_A*_A*k_5*k_5 + 4.*_F*_F + 8.*_A*_F*k_5;
	double m_1 = g_1*k_1;
	double g_2 = 8.*_A*_A*k_4*k_5 + 8.*_A*_D*k_5 + 8.*_A*_F*k_4 + 8.*_D*_F;
	double g_3 = 4.*_A*_A*k_4*k_4 + 4.*_D*_D + 8.*_A*_D*k_4;
	double g_4 = _A*k_1 + _A*k_5*k_5;
	double g_5 = _A*k_2 + 2.*_A*k_4*k_5 + 2.*_D*k_5;
	double g_6 = _A*k_3 + _A*k_4*k_4 + 2.*_D*k_4 + _G;

	//double m_1 = g_1*k_1;
	double m_2 = g_1*k_2 + g_2*k_1;
	double m_3 = g_1*k_3 + g_2*k_2 + g_3*k_1;
	double m_4 = g_2*k_3 + g_3*k_2;
	double m_5 = g_3*k_3;

	double m_6  = _B*_B + 4.*_F*_F*k_5*k_5 + 4.*_B*_F*k_5;
	double m_7  = 4.*_B*_E + 8.*_F*_F*k_4*k_5 + 4.*_B*_F*k_4 + 8.*_E*_F*k_5;
	double m_8  = 4.*_E*_E + 4.*_F*_F*k_4*k_4 + 8.*_E*_F*k_4;
	double m_80 = pow(g_4,2);
	double m_81 = 2*g_4*g_5;
	double m_9  = pow(g_5,2) + 2.*g_4*g_6;
	double m_10 = 2.*g_5*g_6;
	double m_11 = g_6*g_6;

	double m_12 = 	2.*_A*_B*k_1 + 2.*_A*_B*k_5*k_5 + 4.*_A*_F*k_1*k_5 + 4.*_A*_F*pow(k_5,3);
	double m_13 = 	2.*_A*_B*k_2 + 4.*_A*_B*k_4*k_5 + 4.*_B*_D*k_5 + 
		4.*_A*(_E*k_1 + _E*k_5*k_5 + _F*k_1*k_4 + _F*k_2*k_5) + 
		12.*_A*_F*k_4*k_5*k_5 + 8.*_D*_F*k_5*k_5;
	double m_14 = 	2.*_A*_B*k_3 + 2.*_A*_B*k_4*k_4 + 4.*_D*_B*k_4 + 2.*_B*_G + 4.*_A*_E*k_2 + 
		8.*_A*_E*k_4*k_5 + 8.*_E*_D*k_5 + 4.*_A*_F*k_2*k_4 + 4.*_A*_F*k_3*k_5 + 
		12.*_A*_F*k_4*k_4*k_5 + 16.*_D*_F*k_4*k_5 + 4.*_F*_G*k_5;
	double m_15 = 	4.*_A*_E*(k_3 + k_4*k_4) + 8.*_E*_D*k_4 + 4.*_E*_G + 4.*_A*_F*(k_3*k_4 + pow(k_4,3)) + 
		8.*_D*_F*k_4*k_4 + 4.*_F*_G*k_4;

	double  re[5];
	re[0] = m_1 - m_6 - m_12 - m_80;
	re[1] = m_2 - m_7 - m_13 - m_81;
	re[2] = m_3 - m_8 - m_9 - m_14;
	re[3] = m_4 - m_10 - m_15;
	re[4] = m_5 - m_11;  



	/////////////////////////////////////////////////
	double output[8];
	my_qu_kernel(re,output);

	//int sign[2] = {-1, +1}; // NEVER USED
	int ncand(0);

	// deve haver uma solucao mais decente... (devia ser neut..= new std::..
	// mas o new nao funciona em CUDA)
	//std::vector<LorentzVector> n_a;
	//std::vector<LorentzVector> n_b;
	// nao e permitido fazer isto pelo device...
	//vector<myvector> n_c ();
	//vector<LorentzVector> *neutrinoContainer_a = &n_a;
	//vector<LorentzVector> *neutrinoContainer_b = &n_b;

	//////////////////////////////////////////////////
	//// please replace your code starting from here 
	//// ////////////////////////////////////////////


	double rec_x1, rec_y1, rec_z1, rec_e1, rec_x2, rec_y2, rec_z2, rec_e2;
	double m_delta_mass;

	for (int j=0; j<8; j+=2){
		double delta = k_1*output[j]*output[j] + k_2*output[j] + k_3;
//		if ( equal(output[j+1],0) && delta>=0) {
		if ( output[j+1]==0 && delta >=0) {
		//if ( output[j+1]==0 && (k_1*output[j]*output[j] + k_2*output[j] + k_3) >=0) {
			if ( (fx_1 + fx_2*output[j])!=0 ) {
//			if ( nequal((fx_1 + fx_2*output[j]),0) ) {
				rec_x1 = (fx_3*pow(output[j],2) + fx_4*output[j] + fx_5)/(fx_1 + fx_2*output[j]);
			} else {
				rec_x1 = sqrt(delta)+k_4+k_5*output[j];
			}  

			rec_y1 = output[j];
			rec_z1 = G_8/G_7 - G_5*rec_x1/G_7 - G_6*rec_y1/G_7;
			rec_e1 = sqrt(rec_x1*rec_x1 + rec_y1*rec_y1 + rec_z1*rec_z1);
			rec_x2 = S - rec_x1;
			rec_y2 = T - rec_y1;
			rec_z2 = G_12/G_11 - G_9*rec_x2/G_11 - G_10*rec_y2/G_11;
			rec_e2 = sqrt(rec_x2*rec_x2 + rec_y2*rec_y2 + rec_z2*rec_z2);
			
			// self-consistence check and control of the solutions
			double m_w11 = calcMass(rec_x1+lep_a[0], rec_y1+lep_a[1], rec_z1+lep_a[2], rec_e1+lep_a[3]);
			double m_w12 = calcMass(rec_x2+lep_b[0], rec_y2+lep_b[1], rec_z2+lep_b[2], rec_e2+lep_b[3]);
			double m_t11 = calcMass(rec_x1+ bl_a[0], rec_y1+ bl_a[1], rec_z1+ bl_a[2], rec_e1+ bl_a[3]);
			double m_t12 = calcMass(rec_x2+ bl_b[0], rec_y2+ bl_b[1], rec_z2+ bl_b[2], rec_e2+ bl_b[3]);

			m_delta_mass = 1000.0; // allow mass variation range for reco W and tops..
			
			// Avoid thread divergence
			bool m_good_eq1 = ( fabs(S -(rec_x1+rec_x2)) <= 0.01 ) * true + 
							  ( fabs(S -(rec_x1+rec_x2)) > 0.01 ) * false;
			bool m_good_eq2 = ( fabs(T -(rec_y1+rec_y2)) <= 0.01 ) * true +
							  ( fabs(T -(rec_y1+rec_y2)) > 0.01 ) * false;
			bool m_good_eq3 = ( fabs(m_w11 - w_mass[0]) <= m_delta_mass ) * true + 
							  ( fabs(m_w11 - w_mass[0]) > m_delta_mass ) * false;
			bool m_good_eq4 = ( fabs(m_w12 - w_mass[1]) <= m_delta_mass ) * true +
							  ( fabs(m_w12 - w_mass[1]) > m_delta_mass ) * false;
			bool m_good_eq5 = ( fabs(m_t11 - t_mass[0]) <= m_delta_mass ) * true +
							  ( fabs(m_t11 - t_mass[0]) > m_delta_mass ) * false;
			bool m_good_eq6 = ( fabs(m_t12 - t_mass[1]) <= m_delta_mass ) * true +
							  ( fabs(m_t12 - t_mass[1]) <= m_delta_mass ) * false;

			int aux = 2*j;
			bool cond = m_good_eq1 && m_good_eq2 && m_good_eq3 && m_good_eq4 && m_good_eq5 && m_good_eq6;
			syncthreads();
			nc[tid * 16 + aux] = cond * rec_x1;
			nc[tid * 16 + aux + 1] = cond * rec_y1;
			nc[tid * 16 + aux + 2] = cond * rec_z1;
			nc[tid * 16 + aux + 3] = cond * rec_z2;
			ncand += cond * 1;
		}
	}/*
	nc[0] = output[0];
	nc[1] = output[1];
	nc[2] = output[2];
	nc[3] = output[3];
	nc[4] = output[4];
	nc[5] = output[5];
	nc[6] = output[6];
	nc[7] = output[7];*/
	a[tid] = ncand;
	//a[0] = -5;
}  //// end of function

//__host__
//double inline TO1D (double nc[], int tid, int sol, int did)	{
//	return nc[tid*16+sol*4+did];
//}

#define TO1D(nc,tid,sol,did)	nc[tid*16+sol*4+did]

// =========================================
// function that runs the dilep_kernel
// =========================================

	__host__
void runDilep(double t_mass[], double w_mass[], 
		double in_mpx[], double in_mpy[], double in_mpz[],
		TLorentzVector* lep_a, TLorentzVector* lep_b, 
		TLorentzVector* bl_a, TLorentzVector* bl_b, vector<myvector>* &resultfinal)
{
	float time;
	// aloca a memoria no GPU (falta tratar os erros e verificar a capacidade do GPU)
	double *dev_t_mass, *dev_w_mass, *dev_in_mpx, *dev_in_mpy, *dev_in_mpz;
	
//	double ax = lep_a->Px();
//	double ay = lep_a->Py();
//	double az = lep_a->Pz();
//	double ae = lep_a->E();
//	double am = lep_a->M();

	double a[5],b[5],c[5],d[5];
	
	a[0] = lep_a->Px();
	a[1] = lep_a->Py();
	a[2] = lep_a->Pz();
	a[3] = lep_a->E();
	a[4] = lep_a->M();

	b[0] = lep_b->Px();
	b[1] = lep_b->Py();
	b[2] = lep_b->Pz();
	b[3] = lep_b->E();
	b[4] = lep_b->M();

	c[0] = bl_a->Px();
	c[1] = bl_a->Py();
	c[2] = bl_a->Pz();
	c[3] = bl_a->E();
	c[4] = bl_a->M();
     	
	d[0] = bl_b->Px();
	d[1] = bl_b->Py();
	d[2] = bl_b->Pz();
	d[3] = bl_b->E();
	d[4] = bl_b->M();

	double *dev_lep_a, *dev_lep_b, *dev_bl_a, *dev_bl_b;
	double nc[16*NUM_THREADS];
	double *dev_nc;
	int count[NUM_THREADS], *dev_count;
	//resultfinal = new std::vector<myvector>();

	//choseDevice();

// Prepare
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	hipEventRecord(start, 0);

	hipMalloc(&dev_t_mass, 2*sizeof(double));
	hipMalloc(&dev_w_mass, 2*sizeof(double));
	hipMalloc(&dev_in_mpx, 2*sizeof(double));
	hipMalloc(&dev_in_mpy, 2*sizeof(double));
	hipMalloc(&dev_in_mpz, 2*sizeof(double));

	hipMalloc(&dev_lep_a, sizeof(a));
	hipMalloc(&dev_lep_b, sizeof(b));
	hipMalloc(&dev_bl_a, sizeof(c));
	hipMalloc(&dev_bl_b, sizeof(d));
	
	hipMalloc(&dev_nc, 16*NUM_THREADS*sizeof(double));
	hipMalloc(&dev_count, NUM_THREADS*sizeof(int));

	// transfere os dados para a memoria do GPU
	hipMemcpy(dev_t_mass, t_mass, 2*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_w_mass, w_mass, 2*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_in_mpx, in_mpx, 2*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_in_mpy, in_mpy, 2*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_in_mpz, in_mpz, 2*sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(dev_lep_a, &a,	5*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_lep_b, &b,	5*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_bl_a, &c,	5*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_bl_b, &d,	5*sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(dev_nc, nc, 16*NUM_THREADS*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_count, count,NUM_THREADS*sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	total_M1 += elapsedTime;
	//	cout << "Grid " << GRID_SIZE << endl << "Block " << BLOCK_SIZE << endl; 
	// executa a dilep_kernel - 1 dimensao para ser mais facil trabalhar com o array
	dim3 dimGrid(GRID_SIZE, 1);
	dim3 dimBlock(BLOCK_SIZE, 1);

	hipEventRecord(start, 0);
	// todos os blocos fazem o mesmo... 
	dilep_kernel<<<dimGrid,dimBlock>>>(
			dev_t_mass, dev_w_mass, dev_in_mpx, dev_in_mpy, dev_in_mpz, 
			dev_lep_a, dev_lep_b, dev_bl_a, dev_bl_b, dev_nc, dev_count);


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	total_K += elapsedTime;

	hipEventRecord(start, 0);
	// recebe o resultado, nao sei se o nc tambem tem de ser apenas 
	hipMemcpy(nc, dev_nc, 16*NUM_THREADS*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(count, dev_count,NUM_THREADS*sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	total_M2 += elapsedTime;
//	cout << "LOL " << count[0] << endl;
	unsigned int aux_size = GRID_SIZE*BLOCK_SIZE*16, is;

	hipEventDestroy(start);
	hipEventDestroy(stop);
//	cout << resultfinal->size() << " MY_WEIRD_SIZE = " << aux_size;

//	fprintf(flog, "aux_size: %u\n", aux_size);
	for(int thread = 0 ; thread < GRID_SIZE*BLOCK_SIZE ; thread++)	{
		for( int sol = 0 ; sol < count[thread] && sol<4 ; sol++)	{
			


			myvector *mv = new myvector( 
				TO1D(nc,thread,sol,0),
				TO1D(nc,thread,sol,1),
				TO1D(nc,thread,sol,2),
				TO1D(nc,thread,sol,3) );
			
//			cout << endl << "rec_x1: "   << TO1D(nc,thread,sol,0)
//				<< "\trec_y1: " << TO1D(nc,thread,sol,1)
//				<< "\trec_z1: " << TO1D(nc,thread,sol,2)
//				<< "\trec_z2: " << TO1D(nc,thread,sol,3)  << endl;
			
			resultfinal->push_back(*mv);
		}
	}
	
	// i aponta sempre para o inicio do resultado de cada kernel
	// count[i] indica o numero de solucoes desse kernel, cada uma com 4 doubles
/*	for(is = 0; is < aux_size; is += 16){
		for(int aux_j = 0; aux_j < count[is / 16]*4; aux_j += 4){
			
			cout << endl << "rec_x1:" << nc[is+aux_j] << "\trec_y1:" << nc[is+aux_j+1] << " \trec_z1:" << nc[is+aux_j+2] << "\trec_z2" << nc[is+aux_j+3] << endl;
			
			myvector *mv = new myvector(nc[is+aux_j], nc[is+aux_j+1],
					nc[is+aux_j+2], nc[is+aux_j+3]);
			resultfinal->push_back(*mv);
		}
		fprintf(flog,"count: %d\n", count[is/16]);
	}
	fprintf(flog, "sols: %d\n", resultfinal->size());
*/	// faz free das variaveis
	hipFree(dev_t_mass);
	hipFree(dev_w_mass);
	hipFree(dev_in_mpx);
	hipFree(dev_in_mpy);
	hipFree(dev_in_mpz);

	hipFree(dev_lep_a);
	hipFree(dev_lep_b);
	hipFree(dev_bl_a);
	hipFree(dev_bl_b);

	hipFree(dev_count);
	hipFree(dev_nc);

}

// CUDA version must be higher than 2.0 (nao funciona na versao das 8800gt)
// Enough memory to run the kernels
// Highest number of SMPs
// Maximum of 8 GPUs

__host__
int choseDevice(){
	int count, device = -1,  numSMP = 0, sel[8], flag = 0, flag2;
	hipDeviceProp_t properties[8];
	// assumir doubles - ainda nao sei aquilo dos floats
	long int globMem = (30 + 16 * BLOCK_SIZE*GRID_SIZE + BLOCK_SIZE*GRID_SIZE)*8;

	hipGetDeviceCount(&count);
	
	for(int i = 0; i < count && i < 8; i++){

		hipGetDeviceProperties( &properties[i], i);
		
			printf("DEV %s\n", properties[i].name);
		if(strcmp(properties[i].name,"Tesla C2050")==0)	{
			hipSetDevice(i);
			return 0;
		}

		continue;
		
		if(properties[i].major >= 2){
			sel[i] = 1;
			flag++;
		}
		else
			sel[i] = 0;
		
		if(flag == 0){
	//		fprintf(flog, "There is no GPUs capable of running this program on your system.\n");
			return device;
		}

	//	fprintf(flog, "\nNumber of CUDA capable devices: %d\n", count);

		flag = 1;
		for(i = 0; i < count && i < 8; i++){
			if(sel[i]){
				if(properties[i].totalGlobalMem > globMem && properties[i].totalGlobalMem < 8000000000){
					sel[i] += flag;
					flag++;
					globMem = properties[i].totalGlobalMem;
				}
			}
		}
		
		flag2 = 1;

		for(i = 0; i < count && i < 8; i++){
			if(sel[i] == flag){
				if(properties[i].multiProcessorCount > numSMP){
					sel[i] += flag2;
					flag2++;
					numSMP = properties[i].multiProcessorCount;
				}
			}
		}

		for(i = 0; i < count && i < 8; i++){
			if(sel[i] == flag + flag2 - 1){
				hipChooseDevice(&i, &properties[i]);
				hipSetDevice(i);
			//	fprintf(flog, "\n%s GPU chosen\n", properties[i].name);
			//	fprintf(flog, "ID: %d\n", i);
			//	fprintf(flog, "Global Memory: %ld bytes\n", globMem);
			//	fprintf(flog, "Number of SMP: %d\n\n", numSMP);
				return i;
			}
		}
	}
	// Se nao houver nenhuma "Tesla C2050" retorna 1
	return 1;

	return device;
}

//////////////////////////////////////
__device__
void toz_kernel(double k[], double l[], double g[]){
	//// checked !!
	///////////////////////////////////////////////////////////////////////////
	///// bring z=A+Bx+Cy to 2*D*sqrt(x**2+y**2+z**2)-2(ax+by+dz) = E
	///// simplify it to g1*x^2 + g2*y^2 + 2*g3*x + 2*g4*y + 2*g5*xy + g6 = 0
	///////////////////////////////////////////////////////////////////////////
	double A = k[0];
	double B = k[1];
	double C = k[2];
	double D = k[3];
	double E = k[4];
	double a = l[0];
	double b = l[1];
	double d = l[2];
	g[0] = 4*pow(D,2)*( 1 + pow(B,2)) - 4*pow(a,2) -4*pow(d,2)*pow(B,2) - 8*a*d*B;
	if ( g[0]!=0 ) { 
//	if ( nequal(g[0],0) ) { 
		g[1] = ( 4*pow(D,2)*( 1 + pow(C,2)) - 4*pow(b,2) -4*pow(d,2)*pow(C,2) - 8*b*d*C )/g[0] ;
		g[2] = ( (4*pow(D,2)-4*d*d)*A*B - 4*a*d*A - 2*E*a - 2*E*d*B )/g[0];
		g[3] = ( (4*pow(D,2)-4*d*d)*A*C - 4*b*d*A - 2*E*b - 2*E*d*C )/g[0];
		g[4] = ( (4*pow(D,2)-4*d*d)*B*C - 4*a*b   - 4*a*d*C - 4*b*d*B )/g[0];
		g[5] = ( (4*pow(D,2)-4*d*d)*A*A - E*E - 4*E*d*A )/g[0];
		g[0] = 1.0; 
	} else {
		g[1] = ( 4*pow(D,2)*( 1 + pow(C,2)) - 4*pow(b,2) -4*pow(d,2)*pow(C,2) - 8*b*d*C ) ;
		g[2] = ( (4*pow(D,2)-4*d*d)*A*B - 4*a*d*A - 2*E*a - 2*E*d*B );
		g[3] = ( (4*pow(D,2)-4*d*d)*A*C - 4*b*d*A - 2*E*b - 2*E*d*C );
		g[4] = ( (4*pow(D,2)-4*d*d)*B*C - 4*a*b   - 4*a*d*C - 4*b*d*B );
		g[5] = ( (4*pow(D,2)-4*d*d)*A*A - E*E - 4*E*d*A );
		g[0] = 0.;  
	}	 
	return;
}


///////////////////////////////////////////
	__device__
void my_qu_kernel( double my_in[], double my_val[])
{

	///////////////////////////////////////////
	/////  find the solution /////////////////
	/////  ax^4+bx^3+cx^2+dx+e=0
	//////////////////////////////////////////
	double a=my_in[0];
	double b=my_in[1];
	double c=my_in[2];
	double d=my_in[3];
	double e=my_in[4];

	double real[3]={0,0,0};
	double img[3]={0,0,0};
	double x1_r = 0; double x1_i = 0;
	double x2_r = 0; double x2_i = 0;
	double x3_r = 0; double x3_i = 0;
	double x4_r = 0; double x4_i = 0;


	/////////////////////////////////////////////
	///// in case of a==0, simplify to cubic
	///// bx^3+cx^2+dx+e=0
	/////////////////////////////////////////////

	//void cubic(double a[4], double rr[3], double ri[3]);
	if ( a ==0 && b!=0 ){
//	if ( equal(a, 0) && nequal(b,0) ){
		double input[4]={b,c,d,e};
		cubic_kernel(input,real,img);
		x1_r = real[0];	x1_i = img[0];
		x2_r = real[1];	x2_i = img[1];
		x3_r = real[2];	x3_i = img[2];
		my_val[0] = x1_r;
		my_val[1] = x1_i;
		my_val[2] = x2_r;
		my_val[3] = x2_i;
		my_val[4] = x3_r;
		my_val[5] = x3_i;
		my_val[6] = x4_r;
		my_val[7] = x4_i;
		// printf(" the 1st cubic root is: %3.5f %5c %3.5f \n", x1_r, "+i*", x1_i);
		// printf(" the 2nd cubic root is: %3.5f %5c %3.5f \n", x2_r, "+i*", x2_i);
		// printf(" the 3rd cubic root is: %3.5f %5c %3.5f \n", x3_r, "+i*", x3_i);
		return;
	}


	/////////////////////////////////////////////
	///// in case of a==0 && b==0, 
	///// simplify to quadratic
	///// cx*2 + d*x + e =0
	/////////////////////////////////////////////

	if ( a ==0 && b==0 && c!=0){
//	if ( equal(a, 0) && equal(b,0) && nequal(c,0)){
		//double input[3]={c,d,e}; // NEVER USED
		double alpha = pow(d,2)-4*c*e;
		if (alpha>=0) {
			x1_r = (-1*d + sqrt(alpha))/2/c;	x1_i = 0;
			x2_r = (-1*d - sqrt(alpha))/2/c;	x2_i = 0;
		} else {
			x1_r = (-1*d )/2/c;	x1_i = sqrt(-alpha)/2/c;
			x2_r = (-1*d )/2/c;	x2_i = -1*sqrt(-alpha)/2/c;  	
		}
		// printf(" the 1st quadratic root is: %3.5f %5c %3.5f \n", x1_r, "+i*", x1_i);
		// printf(" the 2nd quadratic root is: %3.5f %5c %3.5f \n", x2_r, "+i*", x2_i);
		my_val[0] = x1_r;
		my_val[1] = x1_i;
		my_val[2] = x2_r;
		my_val[3] = x2_i;
		my_val[4] = x3_r;
		my_val[5] = x3_i;
		my_val[6] = x4_r;
		my_val[7] = x4_i;
		return;
	}


	/////////////////////////////////////////////
	///// in case of a==0 && b==0 && c==0, 
	///// simplify to linear equation dx + e =0
	/////////////////////////////////////////////

	if (a ==0 && b==0 && c==0 && d!=0){
//	if (equal(a,0) && equal(b,0) && equal(c,0) && nequal(d,0)){
		x1_r = -e/d;	x1_i = 0;
		my_val[0] = x1_r;
		my_val[1] = x1_i;
		my_val[2] = x2_r;
		my_val[3] = x2_i;
		my_val[4] = x3_r;
		my_val[5] = x3_i;
		my_val[6] = x4_r;
		my_val[7] = x4_i;
		// printf(" the answer to linear equation is: %3.5f %5c %3.5f \n", x1_r, "+i*", x1_i);
		return;	
	}


	//////////////////////////////////////////////
	////  (1)the common expression
	///////////////////////////////////////////////
	// double aa=1; // NEVER USED
	double bb=b/a; double cc=c/a;
	double dd=d/a; double ee=e/a;

	//////////////////////////////////////////////
	////  (2) the equation changes to 
	////	x^4 + bb*x^3 + cc*x^2 + dd*x + ee=0
	////
	////  (3) substitude x=y-aa/4, then we get
	////    y^4 + f*y^2 + g*y + h =0; where
	///////////////////////////////////////////

	double _f = cc - 3*pow(bb,2)/8;
	double _g = dd + (pow(bb,3)/8) - (bb*cc/2);
	double _h = ee - (3*pow(bb,4)/256) + (pow(bb,2)*cc/16) - (bb*dd/4);
	//printf(" _f , _g and _h: %3.5f %3.5f %3.5f \n", _f, _g, _h);

	////////////////////////////////////////////////////////////////////////////
	///  (4) the normal situation is f, g and h are non-zero; then		////
	//// 	the related cubic equation is					////
	//// 	z^3 + (f/2) z^2 + ( (f^2-4h)/16 )*z -g^2/64 =0;			////
	////	 it has three "squared" roots, for example, p,q and l, 		////
	//// 	then p^2, q^2 and l^2 are the root of equation above		////
	//// 	set r=-g/8(pq), then the four roots of the original quartic are	////
	//// 		x = p + q + r -bb/4;					////
	//// 		x = p - q - r -bb/4;					////
	//// 		x = -p + q - r -bb/4;					////
	//// 		x = -p - q + r -bb/4;					////
	////////////////////////////////////////////////////////////////////////////

	double c_1 = 1.;
	double c_2 = _f/2;
	double c_3 = (pow(_f,2)-4*_h)/16.;
	double c_4 = -1*pow(_g,2)/64.;
	//printf(" c_2, c_3 and c_4: %3.5f %3.5f %3.5f\n\n", c_2,c_3,c_4); 

	double input[4]={c_1,c_2,c_3,c_4};
	cubic_kernel(input,real,img);
	//printf(" real part, 1/2/3: %3.5f %3.5f %3.5f \n",real[0],real[1],real[2]);
	//printf(" img part, 1/2/3: %3.5f %3.5f %3.5f \n",img[0],img[1],img[2]);

	////////////////////////////////////////////////
	////// (5) sqrt root of the cubic equation solutions
	////////////////////////////////////////////////

	double out_r[3]={0,0,0};
	double out_i[3]={0,0,0};
	bool _img[3]={false,false,false};
	int img_index =0;
	double my[2];

	for (int ii=0; ii<3; ii++){
		Csqrt_kernel(real[ii],img[ii],my);
		out_r[ii] = my[0];
		out_i[ii] = my[1];
		if ( my[1]!=0 ) {
//		if ( nequal(my[1],0) ) {
			_img[ii]=true;
			img_index++;
		}
		//    cout <<" my "<<out_r[ii]<<" "<<out_i[ii]<< " "<< _img[ii]<<endl;
	}    


	///////////////////////////////////////////////
	////(x + yi)(u + vi) = (xu - yv) + (xv + yu)i
	//// calculating r = -g/(8pq)
	////////////////////////////////////////////////
	double r_r;	double r_i;
	double p_r;	double p_i;
	double q_r;	double q_i;

	if (_img[0]==_img[1] && out_r[0]*out_r[1]>0 ){
//	if (equal(_img[0],_img[1]) && out_r[0]*out_r[1]>0 ){
		r_r	= out_r[0]*out_r[1] - out_i[0]*out_i[1];
		r_i	= out_r[0]*out_i[1] + out_r[1]*out_i[0];
		p_r	= out_r[0]; p_i = out_i[0];
		q_r	= out_r[1]; q_i = out_i[1];
	} else {	
		for (int kk=0; kk<2; kk++) {
			for (int k=kk+1; k<3; k++) {
				if (_img[kk]==_img[k]){ 
//				if (equal(_img[kk],_img[k])){ 
					r_r	= out_r[kk]*out_r[k] - out_i[kk]*out_i[k];
					r_i	= out_r[kk]*out_i[k] + out_r[k]*out_i[kk];
					p_r	= out_r[kk]; p_i = out_i[kk];
					q_r	= out_r[k]; q_i = out_i[k];
					//	cout <<  "taking "<< kk+1 << " and "<< k+1 <<endl;
					//	cout << " kk/k "<< q_r<< " " <<q_i<<" "<<p_r << " "<<p_i<<endl;
				}
			}
		}
	}


	if( r_r !=0 )	{ 
//	if( nequal(r_r, 0))	{ 
		r_r 	= -1.*_g/8/r_r;
	} else {
		r_r 	= 0;
	}
//	if( r_i !=0 )	{ 
//	if( nequal(r_i, 0))	{ 
	if( r_i >= 0.0000001 && r_i <= -0.0000001 )	{ 
		r_r 	= -1.*_g/8/r_i;
	} else {
		r_i 	= 0;
	}	

	//cout << "  rr "<<r_r <<" " <<r_i<<endl;

	x1_r = p_r + q_r + r_r -b/a/4;
	x1_i = p_i + q_i + r_i;
	x2_r = p_r - q_r - r_r -b/a/4;

	x1_r = p_r + q_r + r_r -b/a/4;
	x1_i = p_i + q_i + r_i;
	x2_r = p_r - q_r - r_r -b/a/4;
	x2_i = p_i - q_i - r_i;
	x3_r = -1*p_r + q_r - r_r -b/a/4;
	x3_i = -1*p_i + q_i - r_i;
	x4_r = -1*p_r - q_r + r_r -b/a/4;
	x4_i = -1*p_i - q_i + r_i;

	my_val[0] = x1_r;
	my_val[1] = x1_i;
	my_val[2] = x2_r;
	my_val[3] = x2_i;
	my_val[4] = x3_r;
	my_val[5] = x3_i;
	my_val[6] = x4_r;
	my_val[7] = x4_i;
	/*
	   cout << " 1st root is: "<<x1_r <<" + i*"<<x1_i<<endl;
	   cout << " 2nd root is: "<<x2_r <<" + i*"<<x2_i<<endl;
	   cout << " 3rd root is: "<<x3_r <<" + i*"<<x3_i<<endl;
	   cout << " 4th root is: "<<x4_r <<" + i*"<<x4_i<<endl;
	 */
}
////////////////////end of main
///////////////////////////////////////////////////////////////
////+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	__device__
void Csqrt_kernel(double _ar, double _ai, double _my[])
{
	///// complex sqrt
	//cout << " coming here "<<_ar<<" " <<_ai<<endl; 
	double x,y,r,w;
	if  (  (_ar == 0.0) && (_ai == 0.0) ) {
//	if  (  (equal(_ar , 0.0)) && (equal(_ai , 0.0)) ) {
		_my[0]=0.0;
		_my[1]=0.0;
		return;
	} else {	
		x=fabs(_ar);
		y=fabs(_ai);
		//		cout << x <<" "<<y<<endl;
		if (x >= y) {
			r=y/x;
			w=sqrt(x)*sqrt(0.5*(1.0+sqrt(1.0+r*r)));
		} else {
			r=x/y;
			w=sqrt(y)*sqrt(0.5*(r+sqrt(1.0+r*r)));
		}
		//	cout <<"r/w "<<r<<" "<<w<<endl;
		if (_ar>= 0.0) {
			_my[0]=w;
			_my[1]=_ai/(2.0*w);
		} else {
			_my[1]=(_ai >= 0) ? w : -w;
			_my[0]=_ai/(2.0*_my[1]);
		}
		//		cout << _my[0] <<" " <<_my[1]<<endl;
		return;
	}
}

//////////////////////////////////////////////////////////////////
/// cubic /// a[0]x^3+a[1]x^2+a[2]x+a[3]=0
//////////////////////////////////////////////////////////////////
	__device__
void cubic_kernel(double a[], double rr[], double ri[])
{
	int i;
	double a0, a1, a2, a3;
	double g, h, y1, sh, theta, pi, xy1, xy2, xy3;
	double y2, z1, z2, z3, z4;
	//// initialize the results
	for (i = 0; i < 3; i ++)
	{
		rr[i] = 0.0;
		ri[i] = 0.0;
	}

	a0 = a[0];
	a1 = a[1]/3.0;
	a2 = a[2]/3.0;
	a3 = a[3];

	g = (a0 * a0) * a3 - 3.0 * a0 * a1 * a2 + 2.0 * pow(a1, 3);
	h = a0 * a2 - a1 * a1;
	y1 = g * g + 4.0 * pow(h, 3);

	if (y1 < 0.0){
		sh = sqrt(-h);
	//	xk = 2.0 * sh;
		theta = acos(g / (2.0 * h * sh)) / 3.0;
		xy1 = 2.0 * sh * cos(theta);
		//                pi in double precision
		pi = 3.1415926535897932384626433832795;
		xy2 = 2.0 * sh * cos(theta + (2.0 * pi / 3.0));
		xy3 = 2.0 * sh * cos(theta + (4.0 * pi / 3.0));
		rr[0] = (xy1 - a1) / a0;
		rr[1] = (xy2 - a1) / a0;
		rr[2] = (xy3 - a1) / a0;
		return;
	} else {
		y2 = sqrt(y1);
		z1 = (g + y2) / 2.0;
		z2 = (g - y2) / 2.0;
		if (z1 < 0.0){
			z3 = pow(-z1, 1.0/3.0);
			z3 = -z3;
		} else  z3 = pow(z1, 1.0/3.0);
		if (z2 < 0.0){
			z4 = pow(-z2, 1.0/3.0);
			z4 = - z4;
		}
		else  z4 = pow(z2, 1.0/3.0);

		rr[0] = -(a1 + z3 + z4) / a0;
		rr[1] = (-2.0 * a1 + z3 + z4) / (2.0 * a0);
		ri[1] = sqrt(3.0) * (z4 - z3) / (2.0 * a0);
		rr[2] = rr[1];
		ri[2] = -ri[1];

		return;

	}
}

__device__
 //void _LorentzVectorM(LorentzVector &v);
 double _LorentzVectorM(double x, double y, double z, double e)  {
     double mm=e*e-(x*x+y*y+z*z);
     double res;
	res = sqrt(-mm);

	res *= (mm < 0.0) * -1.0;

	return res;
}

