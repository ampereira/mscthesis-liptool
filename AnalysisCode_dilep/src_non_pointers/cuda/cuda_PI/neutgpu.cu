#include "hip/hip_runtime.h"

#include "neutgpu.cuh"

/*
 *    Using the CUDA Occupancy Calculator it was determined that the optimum
 *    block size is 128.
 *    The best grid size, according to our profiling, is 4096.
 *    This creates a total of 524288 threads, i.e., dilep executions.
 *
 *    The input variance is not implemented. Further profiling is required to
 *    better identify the bottlenecks with the variance.
 *
 *    Note: Avoid float point comparations with absolute values, such as == 0.0,
 *    or f1 == f2. Add a threshold instead.
 *
 *    Note 2: Most of the functions suffered modifications to reduce the amount
 *    of local variables used - reducing register spilling and improving performance - 
 *    at the cost of ilegibility.
 */

unsigned int GRID_SIZE;		// number of blocks per grid
unsigned int BLOCK_SIZE;	// number of threads per block

unsigned int NUM_THREADS;

// Initializes the Grid and Block sizes
__host__ 
void gpu_init(int blocks, int threads)	{
	GRID_SIZE = blocks;
	BLOCK_SIZE = threads;
	NUM_THREADS = blocks*threads;
}

__device__
double calcMass(double x, double y, double z, double e){
	double mm, mass;

	mm = e * e - (x * x + y * y + z * z);

	if(mm < 0.0)
		mass = -sqrt(-mm);
	else
		mass = sqrt(mm);

	return mass;
}

// NEUTRINO SOLUTIONS

// dilep kernel - called from the host (CPU) and executed on the device (GPU)
// Many of the inputs were altered from the CPU version
	__global__
void dilep_kernel(double t_mass[], double w_mass[], double in_mpx[], 
		double in_mpy[], double in_mpz[], double lep_a[], 
		double lep_b[], double bl_a[], double bl_b[], 
		double nc[], int a[])
{

	double G_1, G_3;
	double WMass_a, WMass_b, tMass_a, tMass_b;


	WMass_a = w_mass[0];
	tMass_a = t_mass[0];
	WMass_b = w_mass[1];
	tMass_b = t_mass[1];  
	
	G_1 = (WMass_a - lep_a[4]) * (WMass_a + lep_a[4]);
	G_3 = (WMass_b - lep_b[4]) * (WMass_b + lep_b[4]);

	double G_5,G_6,G_7,G_8,G_9,G_10,G_11,G_12;
	G_5 = ( bl_a[0]/bl_a[3] - lep_a[0]/lep_a[3] );
	G_6 = ( bl_a[1]/bl_a[3] - lep_a[1]/lep_a[3] );
	G_7 = ( bl_a[2]/bl_a[3] - lep_a[2]/lep_a[3] );
	G_8 = ( G_1/lep_a[3] - ((tMass_a - bl_a[4]) * (tMass_a + bl_a[4]))/bl_a[3] )/2.;

	G_9 =	( bl_b[0]/bl_b[3] - lep_b[0]/lep_b[3] );
	G_10 =	( bl_b[1]/bl_b[3] - lep_b[1]/lep_b[3] );
	G_11 =	( bl_b[2]/bl_b[3] - lep_b[2]/lep_b[3] );
	G_12 =	( G_3/lep_b[3] - ((tMass_b - bl_b[4]) * (tMass_b + bl_b[4]))/bl_b[3] )/2.;

	///////////////////////////////////////////////////////////////////
	//// 	G_5 *x1 + G_6*y1 + G_7*z1 = G8;  		(6)
	////  	G_9 *x2 + G_10*y2 + G_11*z2 = G12; 		(7)
	////  	2*El_1*sqrt() - 2*(ax1+by1+cz1) = G_1;  	(8)
	////  	2*El_2*sqrt() - 2*(Ax2+By2+Cz2) = G_3;		(9)
	////  	x1+x2 = S;					(10)
	////  	y1+y2 = T;					(11)
	////  	bring z1 and z2 (from 6/7) to 7 and 8
	///////////////////////////////////////////////////////////////////

	//// 1st top decay product /////
	
	double in_a[5],out_a[6];
	in_a[0] = G_8/G_7;
	in_a[1] = -1.0*G_5/G_7;
	in_a[2] = -1.0*G_6/G_7;
	in_a[3] = lep_a[3];
	in_a[4] = G_1;
	toz_kernel(in_a, lep_a, out_a);

	double in_c[5],out_c[6];
	in_c[0] = G_12/G_11;
	in_c[1] = -1*G_9/G_11;
	in_c[2] = -1*G_10/G_11;
	in_c[3] = lep_b[3];
	in_c[4] = G_3;
	toz_kernel(in_c, lep_b, out_c);
	/////////////////////////////////////////////////////
	//////change x2 y2 equation to x1 and y1 by using
	////// 		x1+x2 = S = in_mpx[0]
	////// 		y1+y2 = T = mpy
	/////////////////////////////////////////////////////
	double out_e[6];
	out_e[0] = out_c[0];
	out_e[1] = out_c[1];
	out_e[2] = -1*( out_c[0]*in_mpx[0] + out_c[2] + out_c[4]*in_mpy[0]);
	out_e[3] = -1*( out_c[1]*in_mpy[0] + out_c[3] + out_c[4]*in_mpx[0]);
	out_e[4] = out_c[4]; 
	out_e[5] =( out_c[0]*in_mpx[0]*in_mpx[0] + out_c[1]*in_mpy[0]*in_mpy[0] + 2*out_c[2]*in_mpx[0] + 2*out_c[3]*in_mpy[0] + out_c[5] + 2*out_c[4]*in_mpx[0]*in_mpy[0]);

	///////////////////////////////////////////////////
	///  solve 
	/// {ax2+by2+2dx+2ey+2fxy+g=0		(12)
	/// {Ax2+By2+2Dx+2Ey+2Fxy+G=0		(13)
	/// out_a[6]: 0   1    2    3    4     5
	/// out_a[6]: a   b    d    e    f     g
	/// out_e[6]: A   B    D    E    F     G
	/// if a!=0, everything is OK.
	///
	/// if a==0, then we can get x2 = f(x,y) from (13)
	/// (12) --> [x2 - f(x,y)] + by2 + ... = 0
	///////////////////////////////////////////////////
	
	double fx_1, fx_2, fx_3, fx_4, fx_5;
	double k_1, k_2, k_3, k_4, k_5;

	// bad organization of the code; diminished register spilling
	if ( out_a[0]!=0  ){
		fx_1 = 2.*(out_e[0]*out_a[2] - out_a[0]*out_e[2]);
		fx_2 = 2.*(out_e[0]*out_a[4] - out_a[0]*out_e[4]);
		fx_3 = out_a[0]*out_e[1] - out_e[0]*out_a[1];
		fx_4 = 2.*(out_e[3]*out_a[0] - out_e[0]*out_a[3]);
		fx_5 = out_a[0]*out_e[5] - out_a[5]*out_e[0];

		k_1 = ( out_a[4]*out_a[4] - out_a[0]*out_a[1] )/out_a[0]/out_a[0];
		k_2 = ( 2.*out_a[2]*out_a[4] - 2.*out_a[0]*out_a[3] )/out_a[0]/out_a[0];
		k_3 = ( out_a[2]*out_a[2]-out_a[0]*out_a[5] )/out_a[0]/out_a[0];
		k_4 = -out_a[2]/out_a[0];
		k_5 = -out_a[4]/out_a[0];
	} else {
		if (out_a[0]==0 && out_e[0]!=0 ) {
			fx_1 = 2.*(out_e[0]*(out_a[2] + out_e[2]/out_e[0]) - out_e[2]);
			fx_2 = 2.*(out_e[0]*(out_a[4] + out_e[4]/out_e[0]) - out_e[4]);
			fx_3 = out_e[1] - out_e[0]*(out_a[1] + out_e[1]/out_e[0]);
			fx_4 = 2.*(out_e[3] - out_e[0]*(out_a[3] + out_e[3]/out_e[0]));
			fx_5 = out_e[5] - (out_a[5] + out_e[5]/out_e[0])*out_e[0];

			k_1 = ( (out_a[4] + out_e[4]/out_e[0])*(out_a[4] + out_e[4]/out_e[0]) - (out_a[1] + out_e[1]/out_e[0]) );
			k_2 = ( 2.*(out_a[2] + out_e[2]/out_e[0])*(out_a[4] + out_e[4]/out_e[0]) - 2.*(out_a[3] + out_e[3]/out_e[0]) );
			k_3 = ( (out_a[2] + out_e[2]/out_e[0])*(out_a[2] + out_e[2]/out_e[0])-(out_a[5] + out_e[5]/out_e[0]) );
			k_4 = -(out_a[2] + out_e[2]/out_e[0]);
			k_5 = -(out_a[4] + out_e[4]/out_e[0]);
		}
	}

	if ( out_a[0]==0 && out_e[0]==0){
		return;
	}


	/////
	///// the part above is 
	///// x = (fx3*y**2 + fx4*y + fx5)/(fx1 + fx2*y)
	///// used to get x value once y is known
	/////
	//// if fx1 + fx2*y == 0, then x is 
	//// x = +/-sqrt(k1*y**2 + k2*y + k3) + (k4 + k5*y)
	////


	double g_1 = 4.*out_e[0]*out_e[0]*k_5*k_5 + 4.*out_e[4]*out_e[4] + 8.*out_e[0]*out_e[4]*k_5;
	double m_1 = g_1*k_1;
	double g_2 = 8.*out_e[0]*out_e[0]*k_4*k_5 + 8.*out_e[0]*out_e[2]*k_5 + 8.*out_e[0]*out_e[4]*k_4 + 8.*out_e[2]*out_e[4];
	double g_3 = 4.*out_e[0]*out_e[0]*k_4*k_4 + 4.*out_e[2]*out_e[2] + 8.*out_e[0]*out_e[2]*k_4;
	double g_4 = out_e[0]*k_1 + out_e[0]*k_5*k_5;
	double g_5 = out_e[0]*k_2 + 2.*out_e[0]*k_4*k_5 + 2.*out_e[2]*k_5;
	double g_6 = out_e[0]*k_3 + out_e[0]*k_4*k_4 + 2.*out_e[2]*k_4 + out_e[5];

	double m_2 = g_1*k_2 + g_2*k_1;
	double m_3 = g_1*k_3 + g_2*k_2 + g_3*k_1;
	double m_4 = g_2*k_3 + g_3*k_2;
	double m_5 = g_3*k_3;

	double m_6  = out_e[1]*out_e[1] + 4.*out_e[4]*out_e[4]*k_5*k_5 + 4.*out_e[1]*out_e[4]*k_5;
	double m_7  = 4.*out_e[1]*out_e[3] + 8.*out_e[4]*out_e[4]*k_4*k_5 + 4.*out_e[1]*out_e[4]*k_4 + 8.*out_e[3]*out_e[4]*k_5;
	double m_8  = 4.*out_e[3]*out_e[3] + 4.*out_e[4]*out_e[4]*k_4*k_4 + 8.*out_e[3]*out_e[4]*k_4;
	double m_80 = pow(g_4,2);
	double m_81 = 2*g_4*g_5;
	double m_9  = pow(g_5,2) + 2.*g_4*g_6;
	double m_10 = 2.*g_5*g_6;
	double m_11 = g_6*g_6;

	double m_12 = 	2.*out_e[0]*out_e[1]*k_1 + 2.*out_e[0]*out_e[1]*k_5*k_5 + 4.*out_e[0]*out_e[4]*k_1*k_5 + 4.*out_e[0]*out_e[4]*pow(k_5,3);
	double m_13 = 	2.*out_e[0]*out_e[1]*k_2 + 4.*out_e[0]*out_e[1]*k_4*k_5 + 4.*out_e[1]*out_e[2]*k_5 + 
		4.*out_e[0]*(out_e[3]*k_1 + out_e[3]*k_5*k_5 + out_e[4]*k_1*k_4 + out_e[4]*k_2*k_5) + 
		12.*out_e[0]*out_e[4]*k_4*k_5*k_5 + 8.*out_e[2]*out_e[4]*k_5*k_5;
	double m_14 = 	2.*out_e[0]*out_e[1]*k_3 + 2.*out_e[0]*out_e[1]*k_4*k_4 + 4.*out_e[2]*out_e[1]*k_4 + 2.*out_e[1]*out_e[5] + 4.*out_e[0]*out_e[3]*k_2 + 
		8.*out_e[0]*out_e[3]*k_4*k_5 + 8.*out_e[3]*out_e[2]*k_5 + 4.*out_e[0]*out_e[4]*k_2*k_4 + 4.*out_e[0]*out_e[4]*k_3*k_5 + 
		12.*out_e[0]*out_e[4]*k_4*k_4*k_5 + 16.*out_e[2]*out_e[4]*k_4*k_5 + 4.*out_e[4]*out_e[5]*k_5;
	double m_15 = 	4.*out_e[0]*out_e[3]*(k_3 + k_4*k_4) + 8.*out_e[3]*out_e[2]*k_4 + 4.*out_e[3]*out_e[5] + 4.*out_e[0]*out_e[4]*(k_3*k_4 + pow(k_4,3)) + 
		8.*out_e[2]*out_e[4]*k_4*k_4 + 4.*out_e[4]*out_e[5]*k_4;

	double  re[5];
	re[0] = m_1 - m_6 - m_12 - m_80;
	re[1] = m_2 - m_7 - m_13 - m_81;
	re[2] = m_3 - m_8 - m_9 - m_14;
	re[3] = m_4 - m_10 - m_15;
	re[4] = m_5 - m_11;  



	double output[8];
	my_qu_kernel(re,output);

	int ncand(0);

	double rec_x1, rec_y1, rec_z1, rec_e1, rec_x2, rec_y2, rec_z2, rec_e2;

	unsigned long tid = threadIdx.x + blockIdx.x * blockDim.x;

	for (int j=0; j<8; j+=2){
		double delta = k_1*output[j]*output[j] + k_2*output[j] + k_3;
		if ( output[j+1]==0 && delta >=0) {
			if ( (fx_1 + fx_2*output[j])!=0 ) {
				rec_x1 = (fx_3*pow(output[j],2) + fx_4*output[j] + fx_5)/(fx_1 + fx_2*output[j]);
			} else {
				rec_x1 = sqrt(delta)+k_4+k_5*output[j];
			}  

			rec_y1 = output[j];
			rec_z1 = G_8/G_7 - G_5*rec_x1/G_7 - G_6*rec_y1/G_7;
			rec_e1 = sqrt(rec_x1*rec_x1 + rec_y1*rec_y1 + rec_z1*rec_z1);
			rec_x2 = in_mpx[0] - rec_x1;
			rec_y2 = in_mpy[0] - rec_y1;
			rec_z2 = G_12/G_11 - G_9*rec_x2/G_11 - G_10*rec_y2/G_11;
			rec_e2 = sqrt(rec_x2*rec_x2 + rec_y2*rec_y2 + rec_z2*rec_z2);
			
			// self-consistence check and control of the solutions

			double m_w11 = calcMass(rec_x1+lep_a[0], rec_y1+lep_a[1], rec_z1+lep_a[2], rec_e1+lep_a[3]);
			double m_w12 = calcMass(rec_x2+lep_b[0], rec_y2+lep_b[1], rec_z2+lep_b[2], rec_e2+lep_b[3]);
			double m_t11 = calcMass(rec_x1+ bl_a[0], rec_y1+ bl_a[1], rec_z1+ bl_a[2], rec_e1+ bl_a[3]);
			double m_t12 = calcMass(rec_x2+ bl_b[0], rec_y2+ bl_b[1], rec_z2+ bl_b[2], rec_e2+ bl_b[3]);

			// m_delta_mass is 1000.0
			bool m_good_eq1 = ( fabs(in_mpx[0] -(rec_x1+rec_x2)) <= 0.01 ) * true + 
							  ( fabs(in_mpx[0] -(rec_x1+rec_x2)) > 0.01 ) * false;
			bool m_good_eq2 = ( fabs(in_mpy[0] -(rec_y1+rec_y2)) <= 0.01 ) * true +
							  ( fabs(in_mpy[0] -(rec_y1+rec_y2)) > 0.01 ) * false;
			bool m_good_eq3 = ( fabs(m_w11 - w_mass[0]) <= 1000.0 ) * true + 
							  ( fabs(m_w11 - w_mass[0]) > 1000.0 ) * false;
			bool m_good_eq4 = ( fabs(m_w12 - w_mass[1]) <= 1000.0 ) * true +
							  ( fabs(m_w12 - w_mass[1]) > 1000.0 ) * false;
			bool m_good_eq5 = ( fabs(m_t11 - t_mass[0]) <= 1000.0 ) * true +
							  ( fabs(m_t11 - t_mass[0]) > 1000.0 ) * false;
			bool m_good_eq6 = ( fabs(m_t12 - t_mass[1]) <= 1000.0 ) * true +
							  ( fabs(m_t12 - t_mass[1]) <= 1000.0 ) * false;

			bool cond = m_good_eq1 && m_good_eq2 && m_good_eq3 && m_good_eq4 && m_good_eq5 && m_good_eq6;
			syncthreads();
			nc[tid * 16 + 2*j] = cond * rec_x1;
			nc[tid * 16 + 2*j + 1] = cond * rec_y1;
			nc[tid * 16 + 2*j + 2] = cond * rec_z1;
			nc[tid * 16 + 2*j + 3] = cond * rec_z2;
			ncand += cond * 1;
		}
	}

	// indicates the number of solutions that this thread found
	a[tid] = ncand;
}

#define TO1D(nc,tid,sol,did)	nc[tid*16+sol*4+did]

// =========================================
// function that runs the dilep_kernel
// =========================================

	__host__
void runDilep(double t_mass[], double w_mass[], 
		double in_mpx[], double in_mpy[], double in_mpz[],
		TLorentzVector* lep_a, TLorentzVector* lep_b, 
		TLorentzVector* bl_a, TLorentzVector* bl_b, vector<myvector>* &resultfinal)
{
	float time;
	double *dev_t_mass, *dev_w_mass, *dev_in_mpx, *dev_in_mpy, *dev_in_mpz;

	double a[5],b[5],c[5],d[5];
	
	a[0] = lep_a->Px();
	a[1] = lep_a->Py();
	a[2] = lep_a->Pz();
	a[3] = lep_a->E();
	a[4] = lep_a->M();

	b[0] = lep_b->Px();
	b[1] = lep_b->Py();
	b[2] = lep_b->Pz();
	b[3] = lep_b->E();
	b[4] = lep_b->M();

	c[0] = bl_a->Px();
	c[1] = bl_a->Py();
	c[2] = bl_a->Pz();
	c[3] = bl_a->E();
	c[4] = bl_a->M();
     	
	d[0] = bl_b->Px();
	d[1] = bl_b->Py();
	d[2] = bl_b->Pz();
	d[3] = bl_b->E();
	d[4] = bl_b->M();

	double *dev_lep_a, *dev_lep_b, *dev_bl_a, *dev_bl_b;
	double nc[16*NUM_THREADS];
	double *dev_nc;
	int count[NUM_THREADS], *dev_count;

	// GPU memory allocation of the inputs and outputs of the dilep kernel
	hipMalloc(&dev_t_mass, 2*sizeof(double));
	hipMalloc(&dev_w_mass, 2*sizeof(double));
	hipMalloc(&dev_in_mpx, 2*sizeof(double));
	hipMalloc(&dev_in_mpy, 2*sizeof(double));
	hipMalloc(&dev_in_mpz, 2*sizeof(double));

	hipMalloc(&dev_lep_a, sizeof(a));
	hipMalloc(&dev_lep_b, sizeof(b));
	hipMalloc(&dev_bl_a, sizeof(c));
	hipMalloc(&dev_bl_b, sizeof(d));
	
	hipMalloc(&dev_nc, 16*NUM_THREADS*sizeof(double));
	hipMalloc(&dev_count, NUM_THREADS*sizeof(int));

	// transfer the inputs to GPU memory
	hipMemcpy(dev_t_mass, t_mass, 2*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_w_mass, w_mass, 2*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_in_mpx, in_mpx, 2*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_in_mpy, in_mpy, 2*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_in_mpz, in_mpz, 2*sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(dev_lep_a, &a,	5*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_lep_b, &b,	5*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_bl_a, &c,	5*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_bl_b, &d,	5*sizeof(double), hipMemcpyHostToDevice);

	//hipMemcpy(dev_nc, nc, 16*NUM_THREADS*sizeof(double), hipMemcpyHostToDevice);
	//hipMemcpy(dev_count, count,NUM_THREADS*sizeof(int), hipMemcpyHostToDevice);

	// define the dimensions of the grid and blocks
	// i.e. the number of times dilep is executed
	dim3 dimGrid(GRID_SIZE, 1);
	dim3 dimBlock(BLOCK_SIZE, 1);

	// dilep kernel call
	dilep_kernel<<<dimGrid,dimBlock>>>(
			dev_t_mass, dev_w_mass, dev_in_mpx, dev_in_mpy, dev_in_mpz, 
			dev_lep_a, dev_lep_b, dev_bl_a, dev_bl_b, dev_nc, dev_count);

	// memory transfer of the results from the GPU
	hipMemcpy(nc, dev_nc, 16*NUM_THREADS*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(count, dev_count,NUM_THREADS*sizeof(int), hipMemcpyDeviceToHost);

	unsigned int aux_size = GRID_SIZE*BLOCK_SIZE*16, is;

	// reconstruction of the normal output of dilep
	for(int thread = 0 ; thread < GRID_SIZE*BLOCK_SIZE ; thread++)	{
		for( int sol = 0 ; sol < count[thread] && sol<4 ; sol++)	{
			
			myvector *mv = new myvector( 
				TO1D(nc,thread,sol,0),
				TO1D(nc,thread,sol,1),
				TO1D(nc,thread,sol,2),
				TO1D(nc,thread,sol,3) );
			
			resultfinal->push_back(*mv);
		}
	}
	
	// frees the memory allocated on GPU
	hipFree(dev_t_mass);
	hipFree(dev_w_mass);
	hipFree(dev_in_mpx);
	hipFree(dev_in_mpy);
	hipFree(dev_in_mpz);

	hipFree(dev_lep_a);
	hipFree(dev_lep_b);
	hipFree(dev_bl_a);
	hipFree(dev_bl_b);

	hipFree(dev_count);
	hipFree(dev_nc);

}

__device__
void toz_kernel(double k[], double l[], double g[]){
	//// checked !!
	///////////////////////////////////////////////////////////////////////////
	///// bring z=A+Bx+Cy to 2*D*sqrt(x**2+y**2+z**2)-2(ax+by+dz) = E
	///// simplify it to g1*x^2 + g2*y^2 + 2*g3*x + 2*g4*y + 2*g5*xy + g6 = 0
	///////////////////////////////////////////////////////////////////////////
	g[0] = 4*pow(k[3],2)*( 1 + pow(k[1],2)) - 4*pow(l[0],2) -4*pow(l[2],2)*pow(k[1],2) - 8*l[0]*l[2]*k[1];
	if ( g[0]!=0 ) { 
		g[1] = ( 4*pow(k[3],2)*( 1 + pow(k[2],2)) - 4*pow(l[1],2) -4*pow(l[2],2)*pow(k[2],2) - 8*l[1]*l[2]*k[2] )/g[0] ;
		g[2] = ( (4*pow(k[3],2)-4*l[2]*l[2])*k[0]*k[1] - 4*l[0]*l[2]*k[0] - 2*k[4]*l[0] - 2*k[4]*l[2]*k[1] )/g[0];
		g[3] = ( (4*pow(k[3],2)-4*l[2]*l[2])*k[0]*k[2] - 4*l[1]*l[2]*k[0] - 2*k[4]*l[1] - 2*k[4]*l[2]*k[2] )/g[0];
		g[4] = ( (4*pow(k[3],2)-4*l[2]*l[2])*k[1]*k[2] - 4*l[0]*l[1]   - 4*l[0]*l[2]*k[2] - 4*l[1]*l[2]*k[1] )/g[0];
		g[5] = ( (4*pow(k[3],2)-4*l[2]*l[2])*k[0]*k[0] - k[4]*k[4] - 4*k[4]*l[2]*k[0] )/g[0];
		g[0] = 1.0; 
	} else {
		g[1] = ( 4*pow(k[3],2)*( 1 + pow(k[2],2)) - 4*pow(l[1],2) -4*pow(l[2],2)*pow(k[2],2) - 8*l[1]*l[2]*k[2] ) ;
		g[2] = ( (4*pow(k[3],2)-4*l[2]*l[2])*k[0]*k[1] - 4*l[0]*l[2]*k[0] - 2*k[4]*l[0] - 2*k[4]*l[2]*k[1] );
		g[3] = ( (4*pow(k[3],2)-4*l[2]*l[2])*k[0]*k[2] - 4*l[1]*l[2]*k[0] - 2*k[4]*l[1] - 2*k[4]*l[2]*k[2] );
		g[4] = ( (4*pow(k[3],2)-4*l[2]*l[2])*k[1]*k[2] - 4*l[0]*l[1]   - 4*l[0]*l[2]*k[2] - 4*l[1]*l[2]*k[1] );
		g[5] = ( (4*pow(k[3],2)-4*l[2]*l[2])*k[0]*k[0] - k[4]*k[4] - 4*k[4]*l[2]*k[0] );
		g[0] = 0.;  
	}	 
	return;
}

	__device__
void my_qu_kernel( double my_in[], double my_val[])
{

	///////////////////////////////////////////
	/////  find the solution /////////////////
	/////  ax^4+bx^3+cx^2+dx+e=0
	//////////////////////////////////////////
	double real[3]={0,0,0};
	double img[3]={0,0,0};
	double x1_r = 0; double x1_i = 0;
	double x2_r = 0; double x2_i = 0;
	double x3_r = 0; double x3_i = 0;
	double x4_r = 0; double x4_i = 0;


	/////////////////////////////////////////////
	///// in case of a==0, simplify to cubic
	///// bx^3+cx^2+dx+e=0
	/////////////////////////////////////////////

	if ( my_in[0] ==0 && my_in[1]!=0 ){
		double input[4]={my_in[1],my_in[2],my_in[3],my_in[4]};
		cubic_kernel(input,real,img);
		x1_r = real[0];	x1_i = img[0];
		x2_r = real[1];	x2_i = img[1];
		x3_r = real[2];	x3_i = img[2];
		my_val[0] = x1_r;
		my_val[1] = x1_i;
		my_val[2] = x2_r;
		my_val[3] = x2_i;
		my_val[4] = x3_r;
		my_val[5] = x3_i;
		my_val[6] = x4_r;
		my_val[7] = x4_i;

		return;
	}


	/////////////////////////////////////////////
	///// in case of a==0 && b==0, 
	///// simplify to quadratic
	///// cx*2 + d*x + e =0
	/////////////////////////////////////////////

	if ( my_in[0] ==0 && my_in[1]==0 && my_in[2]!=0){
		double alpha = pow(my_in[3],2)-4*my_in[2]*my_in[4];
		if (alpha>=0) {
			x1_r = (-1*my_in[3] + sqrt(alpha))/2/my_in[2];	x1_i = 0;
			x2_r = (-1*my_in[3] - sqrt(alpha))/2/my_in[2];	x2_i = 0;
		} else {
			x1_r = (-1*my_in[3] )/2/my_in[2];	x1_i = sqrt(-alpha)/2/my_in[2];
			x2_r = (-1*my_in[3] )/2/my_in[2];	x2_i = -1*sqrt(-alpha)/2/my_in[2];  	
		}

		my_val[0] = x1_r;
		my_val[1] = x1_i;
		my_val[2] = x2_r;
		my_val[3] = x2_i;
		my_val[4] = x3_r;
		my_val[5] = x3_i;
		my_val[6] = x4_r;
		my_val[7] = x4_i;

		return;
	}


	/////////////////////////////////////////////
	///// in case of a==0 && b==0 && c==0, 
	///// simplify to linear equation dx + e =0
	/////////////////////////////////////////////

	if (my_in[0] ==0 && my_in[1]==0 && my_in[2]==0 && my_in[3]!=0){
		x1_r = -my_in[4]/my_in[3];	x1_i = 0;
		my_val[0] = x1_r;
		my_val[1] = x1_i;
		my_val[2] = x2_r;
		my_val[3] = x2_i;
		my_val[4] = x3_r;
		my_val[5] = x3_i;
		my_val[6] = x4_r;
		my_val[7] = x4_i;

		return;	
	}


	//////////////////////////////////////////////
	////  (1)the common expression
	///////////////////////////////////////////////

	//////////////////////////////////////////////
	////  (2) the equation changes to 
	////	x^4 + bb*x^3 + cc*x^2 + dd*x + ee=0
	////
	////  (3) substitude x=y-aa/4, then we get
	////    y^4 + f*y^2 + g*y + h =0; where
	///////////////////////////////////////////

	double _f = my_in[2]/my_in[0] - 3*pow(my_in[1]/my_in[0],2)/8;
	double _g = my_in[3]/my_in[0] + (pow(my_in[1]/my_in[0],3)/8) - (my_in[1]/my_in[0]*my_in[2]/my_in[0]/2);
	double _h = my_in[4]/my_in[0] - (3*pow(my_in[1]/my_in[0],4)/256) + (pow(my_in[1]/my_in[0],2)*my_in[2]/my_in[0]/16) - (my_in[1]/my_in[0]*my_in[3]/my_in[0]/4);

	////////////////////////////////////////////////////////////////////////////
	///  (4) the normal situation is f, g and h are non-zero; then		////
	//// 	the related cubic equation is					////
	//// 	z^3 + (f/2) z^2 + ( (f^2-4h)/16 )*z -g^2/64 =0;			////
	////	 it has three "squared" roots, for example, p,q and l, 		////
	//// 	then p^2, q^2 and l^2 are the root of equation above		////
	//// 	set r=-g/8(pq), then the four roots of the original quartic are	////
	//// 		x = p + q + r -bb/4;					////
	//// 		x = p - q - r -bb/4;					////
	//// 		x = -p + q - r -bb/4;					////
	//// 		x = -p - q + r -bb/4;					////
	////////////////////////////////////////////////////////////////////////////

	double input[4]={1.0, _f/2, (pow(_f,2)-4*_h)/16., -1*pow(_g,2)/64.};
	cubic_kernel(input, real, img);

	////////////////////////////////////////////////
	////// (5) sqrt root of the cubic equation solutions
	////////////////////////////////////////////////

	double out_r[3]={0,0,0};
	double out_i[3]={0,0,0};
	bool _img[3]={false,false,false};
	int img_index =0;
	double my[2];

	for (int ii=0; ii<3; ii++){
		Csqrt_kernel(real[ii],img[ii],my);
		out_r[ii] = my[0];
		out_i[ii] = my[1];

		if ( my[1]!=0 ) {
			_img[ii]=true;
			img_index++;
		}
	}    


	///////////////////////////////////////////////
	////(x + yi)(u + vi) = (xu - yv) + (xv + yu)i
	//// calculating r = -g/(8pq)
	////////////////////////////////////////////////
	double r_r;	double r_i;
	double p_r;	double p_i;
	double q_r;	double q_i;

	if (_img[0]==_img[1] && out_r[0]*out_r[1]>0 ){
		r_r	= out_r[0]*out_r[1] - out_i[0]*out_i[1];
		r_i	= out_r[0]*out_i[1] + out_r[1]*out_i[0];
		p_r	= out_r[0]; p_i = out_i[0];
		q_r	= out_r[1]; q_i = out_i[1];
	} else {	
		for (int kk=0; kk<2; kk++) {
			for (int k=kk+1; k<3; k++) {
				if (_img[kk]==_img[k]){ 
					r_r	= out_r[kk]*out_r[k] - out_i[kk]*out_i[k];
					r_i	= out_r[kk]*out_i[k] + out_r[k]*out_i[kk];
					p_r	= out_r[kk]; p_i = out_i[kk];
					q_r	= out_r[k]; q_i = out_i[k];
				}
			}
		}
	}


	if( r_r !=0 )	{ 
		r_r 	= -1.*_g/8/r_r;
	} else {
		r_r 	= 0;
	}

	if( r_i >= 0.0000001 && r_i <= -0.0000001 )	{ 
		r_r 	= -1.*_g/8/r_i;
	} else {
		r_i 	= 0;
	}	

	x1_r = p_r + q_r + r_r -my_in[1]/my_in[0]/4;
	x1_i = p_i + q_i + r_i;
	x2_r = p_r - q_r - r_r -my_in[1]/my_in[0]/4;

	x1_r = p_r + q_r + r_r -my_in[1]/my_in[0]/4;
	x1_i = p_i + q_i + r_i;
	x2_r = p_r - q_r - r_r -my_in[1]/my_in[0]/4;
	x2_i = p_i - q_i - r_i;
	x3_r = -1*p_r + q_r - r_r -my_in[1]/my_in[0]/4;
	x3_i = -1*p_i + q_i - r_i;
	x4_r = -1*p_r - q_r + r_r -my_in[1]/my_in[0]/4;
	x4_i = -1*p_i - q_i + r_i;

	my_val[0] = x1_r;
	my_val[1] = x1_i;
	my_val[2] = x2_r;
	my_val[3] = x2_i;
	my_val[4] = x3_r;
	my_val[5] = x3_i;
	my_val[6] = x4_r;
	my_val[7] = x4_i;
}

	__device__
void Csqrt_kernel(double _ar, double _ai, double _my[])
{
	///// complex sqrt
	double x,y,r,w;
	if  (  (_ar == 0.0) && (_ai == 0.0) ) {
		_my[0]=0.0;
		_my[1]=0.0;

		return;
	} else {	
		x=fabs(_ar);
		y=fabs(_ai);

		if (x >= y) {
			r=y/x;
			w=sqrt(x)*sqrt(0.5*(1.0+sqrt(1.0+r*r)));
		} else {
			r=x/y;
			w=sqrt(y)*sqrt(0.5*(r+sqrt(1.0+r*r)));
		}

		if (_ar>= 0.0) {
			_my[0]=w;
			_my[1]=_ai/(2.0*w);
		} else {
			_my[1]=(_ai >= 0) ? w : -w;
			_my[0]=_ai/(2.0*_my[1]);
		}

		return;
	}
}

//////////////////////////////////////////////////////////////////
/// cubic /// a[0]x^3+a[1]x^2+a[2]x+a[3]=0
//////////////////////////////////////////////////////////////////
#define pi 3.1415926535897932384626433832795

__device__
void cubic_kernel(double a[], double rr[], double ri[])
{
	int i;
	double a1, a2;
	double g, y1, sh, theta;
	double z1, z2, z3, z4;
	//// initialize the results
	for (i = 0; i < 3; i ++)
	{
		rr[i] = 0.0;
		ri[i] = 0.0;
	}

	a1 = a[1]/3.0;
	a2 = a[2]/3.0;

	g = (a[0] * a[0]) * a[3] - 3.0 * a[0] * a1 * a2 + 2.0 * pow(a1, 3);
	y1 = g * g + 4.0 * pow(a[0] * a2 - a1 * a1, 3);

	if (y1 < 0.0){
		sh = sqrt(-(a[0] * a2 - a1 * a1));
		theta = acos(g / (2.0 * (a[0] * a2 - a1 * a1) * sh)) / 3.0;
		
		rr[0] = (2.0 * sh * cos(theta) - a1) / a[0];
		rr[1] = (2.0 * sh * cos(theta + (2.0 * pi / 3.0)) - a1) / a[0];
		rr[2] = (2.0 * sh * cos(theta + (4.0 * pi / 3.0)) - a1) / a[0];
		return;
	} else {
		z1 = (g + sqrt(y1)) / 2.0;
		z2 = (g - sqrt(y1)) / 2.0;
		if (z1 < 0.0){
			z3 = pow(-z1, 1.0/3.0);
			z3 = -z3;
		} else  z3 = pow(z1, 1.0/3.0);
		if (z2 < 0.0){
			z4 = pow(-z2, 1.0/3.0);
			z4 = - z4;
		}
		else  z4 = pow(z2, 1.0/3.0);

		rr[0] = -(a1 + z3 + z4) / a[0];
		rr[1] = (-2.0 * a1 + z3 + z4) / (2.0 * a[0]);
		ri[1] = sqrt(3.0) * (z4 - z3) / (2.0 * a[0]);
		rr[2] = rr[1];
		ri[2] = -ri[1];

		return;

	}
}
